#include "hip/hip_runtime.h"
#include <iostream>

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "backends/MachineBackend.hpp"
#include "backends/ClassicalBackend.hpp"
#include "backends/AVX512Backend.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

typedef struct RAMImage
{
	uint8_t* data;
	uint32_t size;
} RAMImage;

typedef struct State
{
	uint32_t pc;
	uint32_t x;
} State;

void setup()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for Cuda Fuzzer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for(int i = 0; i < deviceCount; i++)
	{
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

	// First we copy an image of the program and memory to execute
	// Then we initialize the program state
	// Then we launch the kernel

	// It's a bit strange here just because we want to load the program into a buffer on the host side
	// (Along with say, the initial memory) and then we want to copy all that for each instance on
	// the device side

	// Then we want our kernel call to give the relevant information for setting up the initial state of the program

	// Once we have the processor state, and a copy of the memory on the device, we can start executing everything
	
    // hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    // hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);
    // hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    // hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    // hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

	// if(numberOfCircles < 1000)
	// {
	// 	maxAcreSubdiv = 2;
	// }
	
	// int globalHitsSize = ((2 << (maxAcreSubdiv + acreStartdiv)) * (2 << (maxAcreSubdiv + acreStartdiv))) * numberOfCircles;
	// int globalHitCountsSize = (2 << (maxAcreSubdiv + acreStartdiv)) * (2 << (maxAcreSubdiv + acreStartdiv));
        
	// hipError_t mallocErrorCode = hipMalloc(&globalHits, 3 * (sizeof(int) * globalHitsSize) + 2 * (sizeof(int) * globalHitCountsSize));

	// if(mallocErrorCode != hipSuccess)
	// {
	// 	printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(mallocErrorCode));
	// }
    
    // //hipMalloc(&cudaDeviceParts, sizeof(unsigned char) * numberOfCircles);

    // hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    // hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);

    // // Initialize parameters in constant memory.  We didn't talk about
    // // constant memory in class, but the use of read-only constant
    // // memory here is an optimization over just sticking these values
    // // in device global memory.  NVIDIA GPUs have a few special tricks
    // // for optimizing access to constant memory.  Using global memory
    // // here would have worked just as well.  See the Programmer's
    // // Guide for more information about constant memory.

    // GlobalConstants params;
    // params.sceneName = sceneName;
    // params.numberOfCircles = numberOfCircles;
    // params.imageWidth = image->width;
    // params.imageHeight = image->height;
    // params.position = cudaDevicePosition;
    // params.velocity = cudaDeviceVelocity;
    // params.color = cudaDeviceColor;
    // params.radius = cudaDeviceRadius;
    // params.imageData = cudaDeviceImageData;
    
    // //params.parts = cudaDeviceParts;

    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // // Also need to copy over the noise lookup tables, so we can
    // // implement noise on the GPU
    // int* permX;
    // int* permY;
    // float* value1D;
    // getNoiseTables(&permX, &permY, &value1D);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // // Copy over the color table that's used by the shading
    // // function for circles in the snowflake demo

    // float lookupTable[COLOR_MAP_SIZE][3] = {
    //     {1.f, 1.f, 1.f},
    //     {1.f, 1.f, 1.f},
    //     {.8f, .9f, 1.f},
    //     {.8f, .9f, 1.f},
    //     {.8f, 0.8f, 1.f},
    // };

    // hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

__device__ void  runInstruction(State* state, uint32_t inst, uint8_t* memory, uint32_t memorySize)
{	
	uint32_t rd = (inst >> 7) & 0x1f; // Bits 11 to 7

	uint32_t opcode = inst & 0x7f;
	
	// I literally just put these in the order they are in as I read them from page 106 of the
	// RISCV user guide version 2.2 lol
	// There are certainly better ways to do this!
	switch(opcode)
	{
		case 0x37: // lui
		{
			// We don't need to load it into low bits, then reshift it into high bits... can just read the bits in place!
			// Lower bits are filled with zeros according to standard
			state->x[rd] = inst & 0xfffff000;
			state->pc += 4;
			break;
		}
		case 0x17: // auipc
		{
			// Mirrors the above, but result is imm + offset from pc
			state->x[rd] = state->pc + (inst & 0xfffff000);
			state->pc += 4;
			break;
		}
		case 0x6f: // jal
		{
			// This part seems like it would be much nicer in hardware...
			// The bit order is very strange, [20|10:1|11|19:12]
			// so 31 -> 20 == 11, 30 -> 10 == 20, 20 -> 11 == 9, 19 -> 19 == 0
			// Since right shift doing sign extension is implementation dependent, and
			// this wants sign extension, we do it manually...
			// also, yes, this is correct -- it doesn't set lsb
			uint32_t imm = ((inst & (1 << 31)) >> 11) | ((inst & 0x7fe00000) >> 20) | ((inst & 0x00100000) >> 9) | (inst & 0x000ff000);
			state->x[rd] = state->pc + 4;
			// Two cases: either our machine does sign extension and this is redundant, or it defaults to 0 extension and we need this
			// No machine will default to 1 extension so we're all good
			if(inst & (1 << 31))
			{
				imm |= 0xffe00000;
			}
			state->pc += imm;
			break;
		}
		case 0x67: // jalr
		{
			// This wants us to use a temporary in case the destination register and source register are the same
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t temp = state->pc + 4;
			// Oh yeah we have to sign this one again, but bits are nicer, [11:0], so 31 -> 11 == 20
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			state->pc = (state->x[rs1] + imm) & ~1;
			state->x[rd] = temp;
			break;
		}
		case 0x63: // beq, bne, blt, bge, bltu, bgeu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			// The immediate for jump offset is cursed again, high bits are [12|10:5] and then rd has [4:1|11]
			// 31 -> 12 == 19, 30 -> 10 == 20, 4 -> 4 == 0, 0 -> 11 == -11
			// we have to sign extend again as well
			uint32_t imm = ((inst & (1 << 31)) >> 19) | ((inst & 0x7e000000) >> 20) | (rd & 0x1e) | ((rd & 0x1) << 11);
			if(inst & (1 << 31))
			{
				imm |= 0xffffe000;
			}
			// funct3 (bits 14:12) determines which of the comparisons to do
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // beq
				{
					if(state->x[rs1] == state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x1: // bne
				{
					if(state->x[rs1] != state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x4: // blt (this is signed)
				{
					if((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x5: // bge (this is signed)
				{
					if((int32_t)state->x[rs1] >= (int32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x6: // bltu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x7: // bgeu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] >= (uint32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x03: // lb, lh, lw, lbu, lhu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			// Same format as jalr
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // lb
				{
					uint8_t loaded = *(uint8_t*)(memory + (state->x[rs1] + imm));
					state->x[rd] = (loaded & (1 << 7)) ? loaded | 0xffffff00 : loaded;
					break;
				}
				case 0x1: // lh
				{
					uint16_t loaded = *(uint16_t*)(memory + (state->x[rs1] + imm));
					state->x[rd] = (loaded & (1 << 15)) ? loaded | 0xffff0000 : loaded;
					break;
				}
				case 0x2: // lw
				{
					state->x[rd] = *(uint32_t*)(memory + (state->x[rs1] + imm));
					break;
				}
				case 0x4: // lbu
				{
					uint8_t loaded = *(uint8_t*)(memory + (state->x[rs1] + imm));
					state->x[rd] = loaded & 0x000000ff;
					break;
				}
				case 0x5: // lhu
				{
					uint16_t loaded = *(uint16_t*)(memory + (state->x[rs1] + imm));
					state->x[rd] = loaded & 0x0000ffff;
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x23: // sb, sh, sw
		{
			// In this one, we reuse rs1 as the memory location (well plus the immediate offset) and we use rs2 as the source
			// This means the immediate is split up again
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			uint32_t imm = ((inst & 0xfe000000) >> 20) | rd;
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // sb
				{
					*(uint8_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				case 0x1: // sh
				{
					*(uint16_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				case 0x2: // sw
				{
					*(uint32_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				// TODO: handle default?
			}
			state->pc += 4;
			break;
		}
		case 0x13: // addi, slti, sltiu, xori, ori, andi, slli, srli, srai
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // addi
				{
					state->x[rd] = state->x[rs1] + imm;
					break;
				}
				case 0x2: // slti
				{
					// I'm pretty sure c standard says true statements always get set to 1 but just to make
					// it clear
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)imm) ? 1 : 0;
					break;
				}
				case 0x3: // sltiu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)imm) ? 1 : 0;
					break;
				}
				case 0x4: // xori
				{
					state->x[rd] = state->x[rs1] ^ imm;
					break;
				}
				case 0x6: // ori
				{
					state->x[rd] = state->x[rs1] | imm;
					break;
				}
				case 0x7: // andi
				{
					state->x[rd] = state->x[rs1] & imm;
					break;
				}
				case 0x1: // slli
				{
					// TODO: these instructions only use the lowest 5 bits of imm, and
					// the standard says the high bits are all 0 (or 1 of them is 1 for srai)
					// I assume it should be illegal operation if that's not the case?
					state->x[rd] = state->x[rs1] << (imm & 0x1f);
					break;
				}
				case 0x5: // srli, srai are differentiated by a 1 in the 30th bit
				{
					uint32_t shamt = imm & 0x1f;
					if(inst & (1 << 30))
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if((state->x[rs1] & (1 << 31)) && shamt)
						{
							// Bit shifts by 32 are undefined by c standard so we actually can't use this which is extremely cringe
							// because it won't work on 0 shift... so we just special case it. 
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						// Don't do sign extension (don't need to do anything special here)
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x33: // add, sub, sll, slt, sltu, xor, srl, sra, or, and
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // add, sub are differentiated again by funct7 (only 1 bit of it tho), inst bit 30
				{
					// Oh and arithmetic overflow is ignored (aka we don't care, and you know what, just use what our implementation does)
					// This isn't 122
					if(inst & (1 << 30)) // add
					{
						state->x[rd] = state->x[rs1] + state->x[rs2];
					}
					else // sub
					{
						state->x[rd] = state->x[rs1] - state->x[rs2];
					}
					break;
				}
				case 0x1: // sll
				{
					// This only cares about the lower 5 bits
					state->x[rd] = state->x[rs1] << (state->x[rs2] & 0x1f);
					break;
				}
				case 0x2: // slt
				{
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x3: // sltu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x4: // xor
				{
					state->x[rd] = state->x[rs1] ^ state->x[rs2];
					break;
				}
				case 0x5: // srl, sra
				{
					uint32_t shamt = state->x[rs2] & 0x1f;
					if(inst & (1 << 30)) 
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if(state->x[rs1] & (1 << 31) && shamt)
						{
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
				case 0x6: // or
				{
					state->x[rd] = state->x[rs1] | state->x[rs2];
					break;
				}
				case 0x7: // and
				{
					state->x[rd] = state->x[rs1] & state->x[rs2];
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x0f: // fence, fence.i
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
		case 0x73: // ecall, ebreak, csrrw, csrrs, csrrc, csrrwi, csrrsi, csrrci
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
	}

	// We could have written to 0, so just put it back to 0
	if(rd == 0) 
	{
		state->x[rd] = 0;
	}
}

__global__ void kernelExecuteProgram()
{
	State state;
	
	initState(state);
	// We initalize a fake return address so that we can tell when we're done lol
	// Make sure it's 4 byte aligned!
	uint32_t const DONE_ADDRESS = 0xfffffff0; 
	state.x[1] = DONE_ADDRESS;

	// We set the stack pointer to 0 cuz, uh, sure
	state.x[2] = memorySize; 

	while(1)
	{
		uint32_t inst = *(uint32_t*)(program + state.pc);
		printf("executing instruction: %08x\n", inst);
		runInstruction(&state, inst, memory);
		printf("pc = %u\n", state.pc);
		if(state.pc == DONE_ADDRESS)
		{
			break;
		}
	}
	
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

int main(int argc, char** argv)
{
    if (argc != 2) {
        printf("Pass one argument, the filename.\n");
        return 1;
    }

    FILE* programFile = fopen(argv[1], "rb");
    if (!programFile) {
        printf("Couldn't open program file \"%s\".\n", argv[1]);
        return 1;
    }
    fseek(programFile, 0L, SEEK_END); // Technically it wants a long... but
    // The program file cannot possibly be more than can fit in a 32 because it's 32 bit lol
    const auto programSize = ftell(programFile);
    rewind(programFile);

    uint8_t* memory = nullptr;
    uint8_t* program = nullptr;

    memory = static_cast<uint8_t*>(malloc(MEMORY_SIZE + programSize));
    if (!memory) {
        printf("Failed to allocate memory for the emulator.\n");
        return 1;
    }
    program = memory + MEMORY_SIZE;
    fread(program, sizeof(uint8_t), programSize, programFile);

    auto state = State();
    // We initalize a fake return address so that we can tell when we're done lol
    state.x[1] = DONE_ADDRESS;
    // We set the stack pointer to 0 cuz, uh, sure
    state.x[2] = MEMORY_SIZE;

	setup();

    auto backend = ClassicalBackend(memory, state);
    backend.run();

    free(memory);

    return 0;
}
