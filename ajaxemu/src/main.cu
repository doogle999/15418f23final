#include "hip/hip_runtime.h"
#include <iostream>

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <atomic>
#include <random>

#include "mpi.h"

typedef struct State
{
    uint32_t pc;
    uint32_t x[32];
} State;

typedef struct Result
{
    int32_t returnVal;
    int32_t errorCode;
} Result;

typedef struct BranchData
{
    uint32_t hasBeenTaken;
    uint32_t hasBeenSkipped;
} BranchData;

// void setup()
// {
//     int deviceCount = 0;
//     std::string name;
//     hipError_t err = hipGetDeviceCount(&deviceCount);

//     printf("---------------------------------------------------------\n");
//     printf("Initializing CUDA for Cuda Fuzzer\n");
//     printf("Found %d CUDA devices\n", deviceCount);

//     for(int i = 0; i < deviceCount; i++)
// 	{
//         hipDeviceProp_t deviceProps;
//         hipGetDeviceProperties(&deviceProps, i);
//         name = deviceProps.name;

//         printf("Device %d: %s\n", i, deviceProps.name);
//         printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
//         printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
//         printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
//     }
//     printf("---------------------------------------------------------\n");
// }

int classicalExecuteInstruction(State* state, uint32_t inst, uint8_t* memory, uint8_t* program, uint32_t memorySize, uint32_t programSize, BranchData* branchResults)
{
	// Normally this is the destination register, but in S and B type instructions
	// where there is not destination register these same bits communicate parts of an immediate
	// value. We always need to look at these bits as a unit no matter what
	uint32_t rd = (inst >> 7) & 0x1f; // Bits 11 to 7

	uint32_t opcode = inst & 0x7f;
	
	// I literally just put these in the order they are in as I read them from page 106 of the
	// RISCV user guide version 2.2 lol
	// There are certainly better ways to do this!
	switch(opcode)
	{
		case 0x37: // lui
		{
			// We don't need to load it into low bits, then reshift it into high bits... can just read the bits in place!
			// Lower bits are filled with zeros according to standard
			state->x[rd] = inst & 0xfffff000;
			state->pc += 4;
			break;
		}
		case 0x17: // auipc
		{
			// Mirrors the above, but result is imm + offset from pc
			state->x[rd] = state->pc + (inst & 0xfffff000);
			state->pc += 4;
			break;
		}
		case 0x6f: // jal
		{
			// This part seems like it would be much nicer in hardware...
			// The bit order is very strange, [20|10:1|11|19:12]
			// so 31 -> 20 == 11, 30 -> 10 == 20, 20 -> 11 == 9, 19 -> 19 == 0
			// Since right shift doing sign extension is implementation dependent, and
			// this wants sign extension, we do it manually...
			// also, yes, this is correct -- it doesn't set lsb
			uint32_t imm = ((inst & (1 << 31)) >> 11) | ((inst & 0x7fe00000) >> 20) | ((inst & 0x00100000) >> 9) | (inst & 0x000ff000);
			state->x[rd] = state->pc + 4;
			// Two cases: either our machine does sign extension and this is redundant, or it defaults to 0 extension and we need this
			// No machine will default to 1 extension so we're all good
			if(inst & (1 << 31))
			{
				imm |= 0xffe00000;
			}
			state->pc += imm;
			break;
		}
		case 0x67: // jalr
		{
			// This wants us to use a temporary in case the destination register and source register are the same
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t temp = state->pc + 4;
			// Oh yeah we have to sign this one again, but bits are nicer, [11:0], so 31 -> 11 == 20
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			state->pc = (state->x[rs1] + (int32_t)imm) & ~1;
			state->x[rd] = temp;
			break;
		}
		case 0x63: // beq, bne, blt, bge, bltu, bgeu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			// The immediate for jump offset is cursed again, high bits are [12|10:5] and then rd has [4:1|11]
			// 31 -> 12 == 19, 30 -> 10 == 20, 4 -> 4 == 0, 0 -> 11 == -11
			// we have to sign extend again as well
			uint32_t imm = ((inst & (1 << 31)) >> 19) | ((inst & 0x7e000000) >> 20) | (rd & 0x1e) | ((rd & 0x1) << 11);
			if(inst & (1 << 31))
			{
				imm |= 0xffffe000;
			}
			// funct3 (bits 14:12) determines which of the comparisons to do
			int32_t takeBranch = 0;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // beq
				{
					if(state->x[rs1] == state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x1: // bne
				{
					if(state->x[rs1] != state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x4: // blt (this is signed)
				{
					if((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x5: // bge (this is signed)
				{
					if((int32_t)state->x[rs1] >= (int32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x6: // bltu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x7: // bgeu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] >= (uint32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			if(takeBranch)
			{
				uint32_t zero = 0;
				__atomic_compare_exchange_n(&(branchResults[state->pc >> 2].hasBeenTaken), &zero, (uint32_t)1, true, __ATOMIC_RELAXED, __ATOMIC_RELAXED);
				state->pc += (int32_t)imm;
			}
			else
			{
				uint32_t zero = 0;
				__atomic_compare_exchange_n(&(branchResults[state->pc >> 2].hasBeenSkipped), &zero, (uint32_t)1, true, __ATOMIC_RELAXED, __ATOMIC_RELAXED);
				state->pc += 4;
			}
			
			break;
		}
		case 0x03: // lb, lh, lw, lbu, lhu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			// Same format as jalr
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
				//printf("sign extended, %u, %d\n", imm, (int32_t)imm);
			}
			// funct3 again
			uint32_t memOffset = (state->x[rs1] + (int32_t)imm);

			uint32_t funct3 = (inst >> 12) & 0x7;
			uint32_t extra = 0;
			switch(funct3)
			{
				case 0x0: { extra = 0; break; }
				case 0x1: { extra = 1; break; }
				case 0x2: { extra = 3; break; }
				case 0x4: { extra = 0; break; }
				case 0x5: { extra = 1; break; }
			}

			// printf("memOffset, extra, %u, %u\n", memOffset, extra);
			// printf("reg value %u\n", state->x[rs1]);
			
			if(memOffset + extra >= memorySize)
			{
			    state->x[0] = -2;
			    return -2;
			}
			uint8_t* basePtr = memory;
			if(memOffset < programSize)
			{
				if(memOffset + extra >= programSize)
				{
					state->x[0] = -1;
					return -1;
				}
				basePtr = program;
			}
			
			switch(funct3)
			{
				case 0x0: // lb
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 7)) ? loaded | 0xffffff00 : loaded;
					break;
				}
				case 0x1: // lh
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 15)) ? loaded | 0xffff0000 : loaded;
					break;
				}
				case 0x2: // lw
				{
					state->x[rd] = *(uint32_t*)(basePtr + memOffset);
					break;
				}
				case 0x4: // lbu
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x000000ff;
					break;
				}
				case 0x5: // lhu
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x0000ffff;
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x23: // sb, sh, sw
		{
			// In this one, we reuse rs1 as the memory location (well plus the immediate offset) and we use rs2 as the source
			// This means the immediate is split up again
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			uint32_t imm = ((inst & 0xfe000000) >> 20) | rd;
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}

			// printf("Storing value: %u to: %u\n", state->x[rs2], (uint32_t)(state->x[rs1] + (int32_t)imm));
			
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // sb
				{
					*(uint8_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				case 0x1: // sh
				{
					*(uint16_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				case 0x2: // sw
				{
					*(uint32_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				// TODO: handle default?
			}
			state->pc += 4;
			break;
		}
		case 0x13: // addi, slti, sltiu, xori, ori, andi, slli, srli, srai
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // addi
				{
					state->x[rd] = state->x[rs1] + (int32_t)imm;
					break;
				}
				case 0x2: // slti
				{
					// I'm pretty sure c standard says true statements always get set to 1 but just to make
					// it clear
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)imm) ? 1 : 0;
					break;
				}
				case 0x3: // sltiu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)imm) ? 1 : 0;
					break;
				}
				case 0x4: // xori
				{
					state->x[rd] = state->x[rs1] ^ imm;
					break;
				}
				case 0x6: // ori
				{
					state->x[rd] = state->x[rs1] | imm;
					break;
				}
				case 0x7: // andi
				{
					state->x[rd] = state->x[rs1] & imm;
					break;
				}
				case 0x1: // slli
				{
					// TODO: these instructions only use the lowest 5 bits of imm, and
					// the standard says the high bits are all 0 (or 1 of them is 1 for srai)
					// I assume it should be illegal operation if that's not the case?
					state->x[rd] = state->x[rs1] << (imm & 0x1f);
					break;
				}
				case 0x5: // srli, srai are differentiated by a 1 in the 30th bit
				{
					uint32_t shamt = imm & 0x1f;
					if(inst & (1 << 30))
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if((state->x[rs1] & (1 << 31)) && shamt)
						{
							// Bit shifts by 32 are undefined by c standard so we actually can't use this which is extremely cringe
							// because it won't work on 0 shift... so we just special case it. 
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						// Don't do sign extension (don't need to do anything special here)
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x33: // add, sub, sll, slt, sltu, xor, srl, sra, or, and
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // add, sub are differentiated again by funct7 (only 1 bit of it tho), inst bit 30
				{
					// Oh and arithmetic overflow is ignored (aka we don't care, and you know what, just use what our implementation does)
					// This isn't 122
					if((inst & (1 << 30)) == 0) // add
					{
						state->x[rd] = state->x[rs1] + state->x[rs2];
					}
					else // sub
					{
						state->x[rd] = state->x[rs1] - state->x[rs2];
					}
					break;
				}
				case 0x1: // sll
				{
					// This only cares about the lower 5 bits
					state->x[rd] = state->x[rs1] << (state->x[rs2] & 0x1f);
					break;
				}
				case 0x2: // slt
				{
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x3: // sltu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x4: // xor
				{
					state->x[rd] = state->x[rs1] ^ state->x[rs2];
					break;
				}
				case 0x5: // srl, sra
				{
					uint32_t shamt = state->x[rs2] & 0x1f;
					if(inst & (1 << 30)) 
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if(state->x[rs1] & (1 << 31) && shamt)
						{
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
				case 0x6: // or
				{
					state->x[rd] = state->x[rs1] | state->x[rs2];
					break;
				}
				case 0x7: // and
				{
					state->x[rd] = state->x[rs1] & state->x[rs2];
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x0f: // fence, fence.i
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
		case 0x73: // ecall, ebreak, csrrw, csrrs, csrrc, csrrwi, csrrsi, csrrci
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
	}

	// We could have written to 0, so just put it back to 0
	if(rd == 0) 
	{
		state->x[rd] = 0;
	}

	return 0;
}

__device__ __inline__ int executeInstruction(State* state, uint32_t inst, uint8_t* memory, uint8_t* program, uint32_t memorySize, uint32_t programSize, BranchData* branchResults)
{
	// Normally this is the destination register, but in S and B type instructions
	// where there is not destination register these same bits communicate parts of an immediate
	// value. We always need to look at these bits as a unit no matter what
	uint32_t rd = (inst >> 7) & 0x1f; // Bits 11 to 7

	uint32_t opcode = inst & 0x7f;
	
	// I literally just put these in the order they are in as I read them from page 106 of the
	// RISCV user guide version 2.2 lol
	// There are certainly better ways to do this!
	switch(opcode)
	{
		case 0x37: // lui
		{
			// We don't need to load it into low bits, then reshift it into high bits... can just read the bits in place!
			// Lower bits are filled with zeros according to standard
			state->x[rd] = inst & 0xfffff000;
			state->pc += 4;
			break;
		}
		case 0x17: // auipc
		{
			// Mirrors the above, but result is imm + offset from pc
			state->x[rd] = state->pc + (inst & 0xfffff000);
			state->pc += 4;
			break;
		}
		case 0x6f: // jal
		{
			// This part seems like it would be much nicer in hardware...
			// The bit order is very strange, [20|10:1|11|19:12]
			// so 31 -> 20 == 11, 30 -> 10 == 20, 20 -> 11 == 9, 19 -> 19 == 0
			// Since right shift doing sign extension is implementation dependent, and
			// this wants sign extension, we do it manually...
			// also, yes, this is correct -- it doesn't set lsb
			uint32_t imm = ((inst & (1 << 31)) >> 11) | ((inst & 0x7fe00000) >> 20) | ((inst & 0x00100000) >> 9) | (inst & 0x000ff000);
			state->x[rd] = state->pc + 4;
			// Two cases: either our machine does sign extension and this is redundant, or it defaults to 0 extension and we need this
			// No machine will default to 1 extension so we're all good
			if(inst & (1 << 31))
			{
				imm |= 0xffe00000;
			}
			state->pc += imm;
			break;
		}
		case 0x67: // jalr
		{
			// This wants us to use a temporary in case the destination register and source register are the same
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t temp = state->pc + 4;
			// Oh yeah we have to sign this one again, but bits are nicer, [11:0], so 31 -> 11 == 20
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			state->pc = (state->x[rs1] + (int32_t)imm) & ~1;
			state->x[rd] = temp;
			break;
		}
		case 0x63: // beq, bne, blt, bge, bltu, bgeu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			// The immediate for jump offset is cursed again, high bits are [12|10:5] and then rd has [4:1|11]
			// 31 -> 12 == 19, 30 -> 10 == 20, 4 -> 4 == 0, 0 -> 11 == -11
			// we have to sign extend again as well
			uint32_t imm = ((inst & (1 << 31)) >> 19) | ((inst & 0x7e000000) >> 20) | (rd & 0x1e) | ((rd & 0x1) << 11);
			if(inst & (1 << 31))
			{
				imm |= 0xffffe000;
			}
			// funct3 (bits 14:12) determines which of the comparisons to do
			int32_t takeBranch = 0;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // beq
				{
					if(state->x[rs1] == state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x1: // bne
				{
					if(state->x[rs1] != state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x4: // blt (this is signed)
				{
					if((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x5: // bge (this is signed)
				{
					if((int32_t)state->x[rs1] >= (int32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x6: // bltu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				case 0x7: // bgeu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] >= (uint32_t)state->x[rs2]) { takeBranch = 1; }
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			if(takeBranch)
			{
				atomicCAS(&(branchResults[state->pc >> 2].hasBeenTaken), (uint32_t)0, (uint32_t)1);
				state->pc += (int32_t)imm;
			}
			else
			{
				atomicCAS(&(branchResults[state->pc >> 2].hasBeenSkipped), (uint32_t)0, (uint32_t)1);
				state->pc += 4;
			}
			
			break;
		}
		case 0x03: // lb, lh, lw, lbu, lhu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			// Same format as jalr
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
				//printf("sign extended, %u, %d\n", imm, (int32_t)imm);
			}
			// funct3 again
			uint32_t memOffset = (state->x[rs1] + (int32_t)imm);

			uint32_t funct3 = (inst >> 12) & 0x7;
			uint32_t extra = 0;
			switch(funct3)
			{
				case 0x0: { extra = 0; break; }
				case 0x1: { extra = 1; break; }
				case 0x2: { extra = 3; break; }
				case 0x4: { extra = 0; break; }
				case 0x5: { extra = 1; break; }
			}

			// printf("memOffset, extra, %u, %u\n", memOffset, extra);
			// printf("reg value %u\n", state->x[rs1]);
			
			if(memOffset + extra >= memorySize)
			{
			    state->x[0] = -2;
			    return -2;
			}
			uint8_t* basePtr = memory;
			if(memOffset < programSize)
			{
				if(memOffset + extra >= programSize)
				{
					state->x[0] = -1;
					return -1;
				}
				basePtr = program;
			}
			
			switch(funct3)
			{
				case 0x0: // lb
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 7)) ? loaded | 0xffffff00 : loaded;
					break;
				}
				case 0x1: // lh
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 15)) ? loaded | 0xffff0000 : loaded;
					break;
				}
				case 0x2: // lw
				{
					state->x[rd] = *(uint32_t*)(basePtr + memOffset);
					break;
				}
				case 0x4: // lbu
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x000000ff;
					break;
				}
				case 0x5: // lhu
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x0000ffff;
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x23: // sb, sh, sw
		{
			// In this one, we reuse rs1 as the memory location (well plus the immediate offset) and we use rs2 as the source
			// This means the immediate is split up again
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			uint32_t imm = ((inst & 0xfe000000) >> 20) | rd;
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}

			// printf("Storing value: %u to: %u\n", state->x[rs2], (uint32_t)(state->x[rs1] + (int32_t)imm));
			
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // sb
				{
					*(uint8_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				case 0x1: // sh
				{
					*(uint16_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				case 0x2: // sw
				{
					*(uint32_t*)(memory + (uint32_t)(state->x[rs1] + (int32_t)imm)) = state->x[rs2];
					break;
				}
				// TODO: handle default?
			}
			state->pc += 4;
			break;
		}
		case 0x13: // addi, slti, sltiu, xori, ori, andi, slli, srli, srai
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // addi
				{
					state->x[rd] = state->x[rs1] + (int32_t)imm;
					break;
				}
				case 0x2: // slti
				{
					// I'm pretty sure c standard says true statements always get set to 1 but just to make
					// it clear
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)imm) ? 1 : 0;
					break;
				}
				case 0x3: // sltiu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)imm) ? 1 : 0;
					break;
				}
				case 0x4: // xori
				{
					state->x[rd] = state->x[rs1] ^ imm;
					break;
				}
				case 0x6: // ori
				{
					state->x[rd] = state->x[rs1] | imm;
					break;
				}
				case 0x7: // andi
				{
					state->x[rd] = state->x[rs1] & imm;
					break;
				}
				case 0x1: // slli
				{
					// TODO: these instructions only use the lowest 5 bits of imm, and
					// the standard says the high bits are all 0 (or 1 of them is 1 for srai)
					// I assume it should be illegal operation if that's not the case?
					state->x[rd] = state->x[rs1] << (imm & 0x1f);
					break;
				}
				case 0x5: // srli, srai are differentiated by a 1 in the 30th bit
				{
					uint32_t shamt = imm & 0x1f;
					if(inst & (1 << 30))
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if((state->x[rs1] & (1 << 31)) && shamt)
						{
							// Bit shifts by 32 are undefined by c standard so we actually can't use this which is extremely cringe
							// because it won't work on 0 shift... so we just special case it. 
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						// Don't do sign extension (don't need to do anything special here)
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x33: // add, sub, sll, slt, sltu, xor, srl, sra, or, and
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // add, sub are differentiated again by funct7 (only 1 bit of it tho), inst bit 30
				{
					// Oh and arithmetic overflow is ignored (aka we don't care, and you know what, just use what our implementation does)
					// This isn't 122
					if((inst & (1 << 30)) == 0) // add
					{
						state->x[rd] = state->x[rs1] + state->x[rs2];
					}
					else // sub
					{
						state->x[rd] = state->x[rs1] - state->x[rs2];
					}
					break;
				}
				case 0x1: // sll
				{
					// This only cares about the lower 5 bits
					state->x[rd] = state->x[rs1] << (state->x[rs2] & 0x1f);
					break;
				}
				case 0x2: // slt
				{
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x3: // sltu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x4: // xor
				{
					state->x[rd] = state->x[rs1] ^ state->x[rs2];
					break;
				}
				case 0x5: // srl, sra
				{
					uint32_t shamt = state->x[rs2] & 0x1f;
					if(inst & (1 << 30)) 
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if(state->x[rs1] & (1 << 31) && shamt)
						{
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
				case 0x6: // or
				{
					state->x[rd] = state->x[rs1] | state->x[rs2];
					break;
				}
				case 0x7: // and
				{
					state->x[rd] = state->x[rs1] & state->x[rs2];
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x0f: // fence, fence.i
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
		case 0x73: // ecall, ebreak, csrrw, csrrs, csrrc, csrrwi, csrrsi, csrrci
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
	}

	// We could have written to 0, so just put it back to 0
	if(rd == 0) 
	{
		state->x[rd] = 0;
	}

	return 0;
}

__global__ void kernelExecuteProgram(uint8_t* program, uint8_t* globalMemory, uint32_t memorySize, int32_t argc, uint32_t argv, uint32_t programSize, uint32_t entry, Result* globalResults, uint32_t maxOps, BranchData* branchResults)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

	uint8_t* memory = globalMemory + (memorySize * index);

	Result* myResults = globalResults + index;

	State state;

	for(int i = 0; i < 32; i++)
	{
		state.x[i] = 0;
	}

	state.pc = entry;

	uint32_t const DONE_ADDRESS_CUDA = 0xfffffff0;
	
	state.x[1] = DONE_ADDRESS_CUDA;
	state.x[2] = argv;

	state.x[10] = argc;
	state.x[11] = argv;

	int count = 0;
	while(count < maxOps)
	{
		uint32_t inst = *(uint32_t*)(program + state.pc);
		//printf("executing instruction: %08x\n", inst);
		//printf("pc = %u\n", state.pc);
		if(executeInstruction(&state, inst, memory, program, memorySize, programSize, branchResults) || state.pc == DONE_ADDRESS_CUDA)
		{
			break;
		}
		count++;
	}

	// if(index % 1000 == 0)
	// {
	// 	printf("my word was \"%s\" and i completed in %d cycles\n", (memory + *(uint32_t*)(memory + (argv + 4))), count);
	// }

	myResults->returnVal = state.x[10];
	myResults->errorCode = state.x[0]; // If we have an error, just write to x[0] and self destruct out of the loop
}

void classicalExecuteProgram(uint8_t* program, uint8_t* memory, uint32_t memorySize, int32_t argc, uint32_t argv, uint32_t programSize, uint32_t entry, Result* results, uint32_t maxOps, BranchData* branchResults)
{
	State state;
	for(int i = 0; i < 32; i++)
	{
		state.x[i] = 0;
	}
	state.pc = entry;
	uint32_t const DONE_ADDRESS_CLASSICAL = 0xfffffff0;
	
	state.x[1] = DONE_ADDRESS_CLASSICAL;
	state.x[2] = argv;

	state.x[10] = argc;
	state.x[11] = argv;

	int count = 0;
	while(count < maxOps)
	{
		uint32_t inst = *(uint32_t*)(program + state.pc);
		//printf("executing instruction: %08x\n", inst);
		if(classicalExecuteInstruction(&state, inst, memory, program, memorySize, programSize, branchResults) || state.pc == DONE_ADDRESS_CLASSICAL)
		{
			break;
		}
		count++;
	}

	results->returnVal = state.x[10];
    results->errorCode = state.x[0]; // If we have an error, just write to x[0] and self destruct out of the loop
}

int loadToMemory(int argc, char** argv, uint32_t INSTANCE_COUNT, uint32_t MEMORY_SIZE, uint8_t** pout, uint8_t** mout, Result** rout, BranchData** bout, uint32_t* psizeout, int32_t* acout, uint32_t* ssout, uint32_t* epout)
{
	if(argc < 4)
	{
        printf("Format: <program file to execute> <entry address as a number in hex> <args to be passed to subject program (at least 1)>");
        return 1;
    }

	// First step: program instructions
	// Reading the program instructions into a buffer
    FILE* programFile = fopen(argv[1], "rb");
    if(!programFile)
	{
        printf("Couldn't open program file \"%s\".\n", argv[1]);
        return 1;
    }
    fseek(programFile, 0, SEEK_END); 
    uint32_t const programSize = ftell(programFile);
    rewind(programFile);
	uint8_t* program = (uint8_t*)malloc(programSize);
    if(!program)
	{
        printf("Failed to allocate enough memory for the instructions for the emulator.\n");
        return 1;
    }
    fread(program, sizeof(uint8_t), programSize, programFile); // We're offset by 4 so we can force 0 addr to be special
	// At this point, host has the program instructions in memory

	// Second step: we need to initialize the state for the processor. This means setting register 0 to all 0s,
	// setting register 1 to the done address (right after last instruction in program), setting register 1 to the top of the stack,
	// setting register 10 to argc, and setting register 11 to argv. To calculate done address and top of stack, we just need to the
	// size of the program and the size of the argument strings, so that means we need to have the input already
	// We also need to set pc, which is constant across instances. All these things we pass when we invoke the kernel
	
	// Third step: input (we're going to base all of our program variability on argv)
	// So we need to produce images of the arguments to send to the device. This is going to reside just above the instance's stack
	// Basically: every instance needs space for initial stuff + some actual stack memory to execute with
	// Nothing is on the stack to start, we pass argc and argv by setting registers 10 and 11
	// So above the stack we have: actual strings, then pointers to them pointed to by argv, then the actual stack
	// So now we allocate the memory images for the program
	uint8_t* memory = (uint8_t*)malloc(MEMORY_SIZE * INSTANCE_COUNT);
	if(!memory)
	{
		printf("Failed to allocate enough memory for the emulator.\n");
		return 1;
	}
	memset(memory, 0, MEMORY_SIZE * INSTANCE_COUNT);
	// For now, we're literally just going to pass through arguments from our actual call of this program.
	// So argv[3..] correspond to argv[1..] in the subject program and argv[1] in our program is argv[0] in subject
	int32_t argcSubj = argc - 2;
	uint32_t* argvSubjOffsets = (uint32_t*)malloc(argcSubj * sizeof(uint32_t));
	argvSubjOffsets[0] = strlen(argv[1]) + 1;
	strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[0])), argv[1], argvSubjOffsets[0]);
	for(int32_t i = 1; i < argcSubj; i++)
	{
		// Can't use stpcpy because we need to know size before hand because we are storing "backwards" because we only know
		// Higher address because stack grows down
		uint32_t tempLength = strlen(argv[i + 2]) + 1;
	    argvSubjOffsets[i] = tempLength + argvSubjOffsets[i - 1];
		if(argvSubjOffsets[i] > MEMORY_SIZE)
		{
			printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
			return 1;
		}
		strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[i])), argv[i + 2], tempLength);
	}
	
	// Still need to copy the pointers to these
	uint32_t argvArrayEnd = argvSubjOffsets[argcSubj - 1];
	argvArrayEnd = argvArrayEnd + ((4 - (argvArrayEnd % 4)) % 4); // Alignment...
	if(argvArrayEnd + (4 * argcSubj) >= MEMORY_SIZE)
	{
		printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
		return 1;
	}
	for(int32_t i = 0; i < argcSubj; i++)
	{
		// All programs see their memory as offset relative to their own memory chunk so this is ok to copy
		*(uint32_t*)(memory + (MEMORY_SIZE - argvArrayEnd - (4 * (i + 1)))) = MEMORY_SIZE - argvSubjOffsets[argcSubj - i - 1];
	}
	// Now all args are copied to the first instances host memory, so we copy them to all the instances
	uint32_t stackStart = MEMORY_SIZE - (argvArrayEnd + (argcSubj * 4)); // Remember, starting stack pointer value is not usable immediately, dec first, so this ok

	char randBuf[32];
	uint32_t maxIn = (argvSubjOffsets[1] - argvSubjOffsets[0]);
	if(maxIn > 31)
	{
		maxIn = 31;
	}
	randBuf[maxIn] = '\0';
    srand(time(NULL));

	for(uint32_t i = 1; i < INSTANCE_COUNT; i++)
	{
		// Make sure memory size is big enough or problems will happen
		memcpy(memory + ((MEMORY_SIZE * i) + stackStart), memory + stackStart, MEMORY_SIZE - stackStart);

		for(int j = 0; j < maxIn; j++)
		{
			randBuf[j] = (rand() % 26) + 97;
		}
		strncpy((char*)(memory + (MEMORY_SIZE * i) + (MEMORY_SIZE - argvSubjOffsets[1])), randBuf, (argvSubjOffsets[1] - argvSubjOffsets[0]));
	}
	free(argvSubjOffsets);
	// Should now have both program and memory images on the device

	uint32_t entryPoint = (uint32_t)strtol(argv[2], NULL, 16);

	uint32_t programInstCount = programSize / 4;

	Result* localResults = (Result*)malloc(INSTANCE_COUNT * sizeof(Result));
	BranchData* localBranchData = (BranchData*)malloc(programInstCount * sizeof(BranchData));
	if(!localResults || !localBranchData)
	{
		printf("FAILED TO malloc results\n");
		return 1;
	}
	memset(localBranchData, 0, sizeof(BranchData) * programInstCount);

	*pout = program;
	*mout = memory;
	*rout = localResults;
	*bout = localBranchData;
	*psizeout = programSize;
	*acout = argcSubj;
	*ssout = stackStart;
	*epout = entryPoint;

	return 0;
}

int main(int argc, char** argv)
{
	int pid;
	int nproc;
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &pid);
    MPI_Comm_size(MPI_COMM_WORLD, &nproc);
    MPI_Comm_set_errhandler(MPI_COMM_WORLD, MPI_ERRORS_ARE_FATAL);
    MPI_Barrier(MPI_COMM_WORLD);

	uint8_t* program;
	uint8_t* memory;
	Result* localResults;
	BranchData* localBranchData;
	uint32_t programSize;
	int32_t argcSubj;
	uint32_t stackStart;
	uint32_t entryPoint;

	uint8_t* deviceProgramImage;
	uint8_t* deviceMemoryImage;
	Result* deviceResultImage;
	BranchData* deviceBranchDataImage;

	uint8_t* spareMemory;

	dim3 blockDim(256);
	dim3 gridDim(16);

	uint32_t const MAX_OPS = 10000;	
	uint32_t const MEMORY_SIZE = 4 * 1024; // This needs to be 4 byte aligned or bad things happen because cuda memory access rules
	uint32_t INSTANCE_COUNT = 1;
	if(pid == 0)
	{
		INSTANCE_COUNT = blockDim.x * gridDim.x;

		if(loadToMemory(argc, argv, INSTANCE_COUNT, MEMORY_SIZE, &program, &memory, &localResults, &localBranchData, &programSize, &argcSubj, &stackStart, &entryPoint))
		{
			return 1;
		}
		hipError_t programMallocErrorCode = hipMalloc(&deviceProgramImage, programSize);
		if(programMallocErrorCode != hipSuccess)
		{
			printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(programMallocErrorCode));
			return 1;
		}

		hipError_t mallocMemoryImageError = hipMalloc(&deviceMemoryImage, MEMORY_SIZE * INSTANCE_COUNT);
		if(mallocMemoryImageError != hipSuccess)
		{
			printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(mallocMemoryImageError));
			return 1;
		}

		hipError_t mallocResultImageError = hipMalloc(&deviceResultImage, INSTANCE_COUNT * sizeof(Result));
		if(mallocResultImageError != hipSuccess)
		{
			printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(mallocResultImageError));
			return 1;
		}
		uint32_t programInstCount = programSize / 4;
		hipError_t mallocBranchDataImageError = hipMalloc(&deviceBranchDataImage, sizeof(BranchData) * (programInstCount));
		if(mallocBranchDataImageError != hipSuccess)
		{
			printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(mallocBranchDataImageError));
			return 1;
		}
		hipMemcpy(deviceProgramImage, program, programSize, hipMemcpyHostToDevice);
		hipMemcpy(deviceBranchDataImage, localBranchData, sizeof(BranchData) * (programInstCount), hipMemcpyHostToDevice);
		hipMemcpy(deviceMemoryImage, memory, MEMORY_SIZE * INSTANCE_COUNT, hipMemcpyHostToDevice);
	}
	else
	{
		if(loadToMemory(argc, argv, INSTANCE_COUNT, MEMORY_SIZE, &program, &memory, &localResults, &localBranchData, &programSize, &argcSubj, &stackStart, &entryPoint))
		{
			return 1;
		}

		spareMemory = (uint8_t*)malloc(MEMORY_SIZE * INSTANCE_COUNT);
		memcpy((spareMemory + stackStart), memory + stackStart, MEMORY_SIZE - stackStart);
	}

	int goodToGo = 1;
	MPI_Barrier(MPI_COMM_WORLD);
	auto startTime = std::chrono::high_resolution_clock::now();

	MPI_Request doneReq;

uint32_t argv1Len = 0; 

    uint32_t maxIn = 0; 

	if(pid != 0)
	{
		MPI_Irecv(&goodToGo,
				  1,
				  MPI_INT,
				  0,
				  0,
				  MPI_COMM_WORLD,
				  &doneReq);

		argv1Len = strlen((char*)(spareMemory + *(uint32_t*)(spareMemory + stackStart + 4)));
		maxIn = argv1Len;
		if(maxIn > 31)
		{
			maxIn = 31;
		}
	}

	uint64_t instancesRun = 0;

	while(goodToGo)
	{
		if(pid == 0)
		{
			//hipMemcpy(deviceMemoryImage, memory, MEMORY_SIZE * INSTANCE_COUNT, hipMemcpyHostToDevice);
			
			kernelExecuteProgram<<<gridDim, blockDim>>>(deviceProgramImage, deviceMemoryImage, MEMORY_SIZE, argcSubj, stackStart, programSize, entryPoint, deviceResultImage, MAX_OPS, deviceBranchDataImage);

			hipError_t errorCode = hipPeekAtLastError();
			if(errorCode != hipSuccess)
			{
		        printf("FAILED TO LAUNCH KERNEL: %s\n", hipGetErrorString(errorCode));
	        }
			hipDeviceSynchronize();

			goodToGo = 0;
			for(int i = 1; i < nproc; i++)
			{
				MPI_Send(&goodToGo,
						 1,
						 MPI_INT,
						 i,
						 0,
						 MPI_COMM_WORLD);
			}
		}
		else
		{
			char randBuf[32];
			randBuf[maxIn] = '\0';
			for(int i = 0; i < maxIn; i++)
			{
				randBuf[i] = (rand() % 26) + 97;
			}
			
			for(int i = 0; i < INSTANCE_COUNT; i++)
			{
				memcpy(memory + ((MEMORY_SIZE * i) + stackStart), spareMemory + stackStart, MEMORY_SIZE - stackStart);
				// This is jsut beautiful -- we don't need to recalculate where argv[1] is because we have the stack LMAO
				char* argv1 = (char*)(memory + (MEMORY_SIZE * i) + *(uint32_t*)(spareMemory + stackStart + 4));
				strncpy(argv1, randBuf, maxIn);
			}
			
		    classicalExecuteProgram(program, memory, MEMORY_SIZE, argcSubj, stackStart, programSize, entryPoint, localResults, MAX_OPS, localBranchData);

			int flag = 0;
			MPI_Test(&doneReq, &flag, MPI_STATUS_IGNORE);
	    }
		instancesRun += INSTANCE_COUNT;
	}

	uint64_t totalInstancesRun = 0;
	MPI_Allreduce(&instancesRun, &totalInstancesRun, 1, MPI_UINT64_T, MPI_SUM, MPI_COMM_WORLD);
	
	auto midExecTime = std::chrono::high_resolution_clock::now();
	MPI_Barrier(MPI_COMM_WORLD);

	if(pid == 0)
	{
		hipMemcpy(localBranchData, deviceBranchDataImage, sizeof(BranchData) * (programSize / 4), hipMemcpyDeviceToHost);
	}

	// for(uint32_t i = 0; i < (programSize / 4); i++)
	// {
	// 	if((((uint32_t*)program)[i] & 0x7f) == 0x63)
	// 	{
	// 		printf("pid %d, Branch at address %x was", pid, i * 4);
	// 		if(localBranchData[i].hasBeenTaken) printf(" taken");
	// 		if(localBranchData[i].hasBeenSkipped) printf(" skipped");
	// 		printf("\n");
	// 	}
	// }

	MPI_Allreduce(MPI_IN_PLACE, localBranchData, 2 * (programSize / 4), MPI_UINT32_T, MPI_BOR, MPI_COMM_WORLD);
	auto finishTime = std::chrono::high_resolution_clock::now();
    printf("pid %d, Exec took %lu us, full + comm took %lu us, ran %lu instances\n", pid, std::chrono::duration_cast<std::chrono::microseconds>(midExecTime - startTime).count(), std::chrono::duration_cast<std::chrono::microseconds>(finishTime - startTime).count(), instancesRun);

	MPI_Barrier(MPI_COMM_WORLD);
	
	if(pid == 0)
	{
	    printf("Total of %lu instances run across %d processes, 1 of which used the gpu\n", totalInstancesRun, nproc);
		for(uint32_t i = 0; i < (programSize / 4); i++)
		{
			if((((uint32_t*)program)[i] & 0x7f) == 0x63)
			{
				printf("Branch at address %x was", i * 4);
				if(localBranchData[i].hasBeenTaken) printf(" taken");
				if(localBranchData[i].hasBeenSkipped) printf(" skipped");
				printf("\n");
			}
		}
	}

	MPI_Barrier(MPI_COMM_WORLD);
	
	if(pid == 0)
	{
		hipFree(deviceProgramImage);
		hipFree(deviceMemoryImage);
		hipFree(deviceResultImage);
		hipFree(deviceBranchDataImage);
	}
	else
	{		
		free(spareMemory);
	}

	free(memory);
	free(program);
	free(localResults);
	free(localBranchData);

	MPI_Finalize();

    return 0;

	// At this point, device has the program instructions in memory

	// Second step: we need to initialize the state for the processor. This means setting register 0 to all 0s,
	// setting register 1 to the done address (right after last instruction in program), setting register 1 to the top of the stack,
	// setting register 10 to argc, and setting register 11 to argv. To calculate done address and top of stack, we just need to the
	// size of the program and the size of the argument strings, so that means we need to have the input already
	// We also need to set pc, which is constant across instances. All these things we pass when we invoke the kernel
	
	// Third step: input (we're going to base all of our program variability on argv)
	// So we need to produce images of the arguments to send to the device. This is going to reside just above the instance's stack
	// Basically: every instance needs space for initial stuff + some actual stack memory to execute with
	// Nothing is on the stack to start, we pass argc and argv by setting registers 10 and 11
	// So above the stack we have: actual strings, then pointers to them pointed to by argv, then the actual stack
	// So now we allocate the memory images for the program
	// uint8_t* memory = (uint8_t*)malloc(MEMORY_SIZE * INSTANCE_COUNT);
	// if(!memory)
	// {
	// 	printf("Failed to allocate enough memory for the emulator.\n");
	// 	return 1;
	// }
	// memset(memory, 0, MEMORY_SIZE * INSTANCE_COUNT);
	// // For now, we're literally just going to pass through arguments from our actual call of this program.
	// // So argv[3..] correspond to argv[1..] in the subject program and argv[1] in our program is argv[0] in subject
	// int32_t argcSubj = argc - 2;
	// uint32_t* argvSubjOffsets = (uint32_t*)malloc(argcSubj * sizeof(uint32_t));
	// argvSubjOffsets[0] = strlen(argv[1]) + 1;
	// strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[0])), argv[1], argvSubjOffsets[0]);
	// for(int32_t i = 1; i < argcSubj; i++)
	// {
	// 	// Can't use stpcpy because we need to know size before hand because we are storing "backwards" because we only know
	// 	// Higher address because stack grows down
	// 	uint32_t tempLength = strlen(argv[i + 2]) + 1;
	//     argvSubjOffsets[i] = tempLength + argvSubjOffsets[i - 1];
	// 	if(argvSubjOffsets[i] > MEMORY_SIZE)
	// 	{
	// 		printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
	// 		return 1;
	// 	}
	// 	strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[i])), argv[i + 2], tempLength);
	// }
	
	// // Still need to copy the pointers to these
	// uint32_t argvArrayEnd = argvSubjOffsets[argcSubj - 1];
	// argvArrayEnd = argvArrayEnd + ((4 - (argvArrayEnd % 4)) % 4); // Alignment...
	// if(argvArrayEnd + (4 * argcSubj) >= MEMORY_SIZE)
	// {
	// 	printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
	// 	return 1;
	// }
	// for(int32_t i = 0; i < argcSubj; i++)
	// {
	// 	// All programs see their memory as offset relative to their own memory chunk so this is ok to copy
	// 	*(uint32_t*)(memory + (MEMORY_SIZE - argvArrayEnd - (4 * (i + 1)))) = MEMORY_SIZE - argvSubjOffsets[argcSubj - i - 1];
	// }
	// // Now all args are copied to the first instances host memory, so we copy them to all the instances
	// uint32_t stackStart = MEMORY_SIZE - (argvArrayEnd + (argcSubj * 4)); // Remember, starting stack pointer value is not usable immediately, dec first, so this ok
	// char randBuf[4];
	// randBuf[3] = '\0';
	// srand(pid * 43214 + 321);
	// for(uint32_t i = 1; i < INSTANCE_COUNT; i++)
	// {
	// 	// Make sure memory size is big enough or problems will happen
	// 	memcpy(memory + ((MEMORY_SIZE * i) + stackStart), memory + stackStart, MEMORY_SIZE - stackStart);

	// 	randBuf[0] = (rand() % 26) + 97;
	// 	randBuf[1] = (rand() % 26) + 97;
	// 	randBuf[2] = (rand() % 26) + 97;
	// 	strncpy((char*)(memory + (MEMORY_SIZE * i) + (MEMORY_SIZE - argvSubjOffsets[1])), randBuf, (argvSubjOffsets[1] - argvSubjOffsets[0]));
	// }
	// // Finally can copy all of them to device... a little wasteful, since much of this will be zeroes, but I figure better than many small calls
	// // could theoretically seperate these regions of memory but would require complex redirect system on emulator memory system...

	// free(argvSubjOffsets);
	// // Should now have both program and memory images on the device

	// uint32_t entryPoint = (uint32_t)strtol(argv[2], NULL, 16);

	// // Oh, and we have to set up a place for return values + important codes

	// Result* localResults = (Result*)malloc(INSTANCE_COUNT * sizeof(Result));
	// BranchData* localBranchData = (BranchData*)malloc(programInstCount * sizeof(BranchData));
	// if(!localResults || !localBranchData)
	// {
	// 	printf("FAILED TO malloc results\n");
	// 	return 1;
	// }

	// auto startTime = std::chrono::high_resolution_clock::now();
	// //



	// auto finishTime = std::chrono::high_resolution_clock::now();

	// printf("Kernel took %lu us\n", std::chrono::duration_cast<std::chrono::microseconds>(finishTime - startTime).count());

	// // Print results
	// //hipMemcpy(localResults, deviceResultImage, sizeof(Result) * INSTANCE_COUNT, hipMemcpyDeviceToHost);

	// for(uint32_t i = 0; i < INSTANCE_COUNT; i++)
	// {
	// 	printf("Instance %u: return %d, errorCode %d\n", i, localResults[i].returnVal, localResults[i].errorCode);
	// }
	// free(localResults);

	// //hipMemcpy(localBranchData, deviceBranchDataImage, sizeof(BranchData) * programInstCount, hipMemcpyDeviceToHost);

	// for(uint32_t i = 0; i < programInstCount; i++)
	// {
	// 	if((((uint32_t*)program)[i] & 0x7f) == 0x63)
	// 	{
	// 		printf("Branch at address %x was", i * 4);
	// 		if(localBranchData[i].hasBeenTaken) printf(" taken");
	// 		if(localBranchData[i].hasBeenSkipped) printf(" skipped");
	// 		printf("\n");
	// 	}
	// }

    // MPI_Barrier(MPI_COMM_WORLD);
	// printf("NOW ALL REDUCING\n");

	// MPI_Allreduce(MPI_IN_PLACE, localBranchData, 2 * programInstCount, MPI_UINT32_T, MPI_BOR, MPI_COMM_WORLD);

	// for(uint32_t i = 0; i < programInstCount; i++)
	// {
	// 	if((((uint32_t*)program)[i] & 0x7f) == 0x63)
	// 	{
	// 		printf("Branch at address %x was", i * 4);
	// 		if(localBranchData[i].hasBeenTaken) printf(" taken");
	// 		if(localBranchData[i].hasBeenSkipped) printf(" skipped");
	// 		printf("\n");
	// 	}
	// }
	// free(localBranchData);

	// // Printing memory dumps
	// //hipMemcpy(memory, deviceMemoryImage, sizeof(uint8_t) * MEMORY_SIZE * INSTANCE_COUNT, hipMemcpyDeviceToHost);

	// // uint32_t const BYTES_PER_LINE = 4 * 4;
	// // for(uint32_t j = 0; j < INSTANCE_COUNT; j++)
	// // {
	// // 	for(uint32_t i = 0; i < MEMORY_SIZE - programSize; i += 1)
	// // 	{
	// // 		if(MEMORY_SIZE - i == stackStart)
	// // 		{
	// // 			printf("\n");
	// // 		}
	// // 		if(i % BYTES_PER_LINE == 0)
	// // 		{
	// // 			printf("\n");
	// // 		}
	// // 		printf("%02x ", *(uint8_t*)(memory + (j * MEMORY_SIZE) + MEMORY_SIZE - i - 1));
	// // 	}
	// // 	printf("\n");
	// // }
   
	
    // free(memory);
	// free(program);

}
