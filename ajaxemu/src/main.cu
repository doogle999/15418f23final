#include <iostream>

#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <hip/hip_runtime.h>


typedef struct State
{
    uint32_t pc;
    uint32_t x[32];
} State;

// void setup()
// {
//     int deviceCount = 0;
//     std::string name;
//     cudaError_t err = cudaGetDeviceCount(&deviceCount);

//     printf("---------------------------------------------------------\n");
//     printf("Initializing CUDA for Cuda Fuzzer\n");
//     printf("Found %d CUDA devices\n", deviceCount);

//     for(int i = 0; i < deviceCount; i++)
// 	{
//         cudaDeviceProp deviceProps;
//         cudaGetDeviceProperties(&deviceProps, i);
//         name = deviceProps.name;

//         printf("Device %d: %s\n", i, deviceProps.name);
//         printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
//         printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
//         printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
//     }
//     printf("---------------------------------------------------------\n");
// }

__device__ __inline__ void executeInstruction(State* state, uint32_t inst, uint8_t* memory, uint8_t* program, uint32_t memorySize, uint32_t programSize)
{
	// Normally this is the destination register, but in S and B type instructions
	// where there is not destination register these same bits communicate parts of an immediate
	// value. We always need to look at these bits as a unit no matter what
	uint32_t rd = (inst >> 7) & 0x1f; // Bits 11 to 7

	uint32_t opcode = inst & 0x7f;
	
	// I literally just put these in the order they are in as I read them from page 106 of the
	// RISCV user guide version 2.2 lol
	// There are certainly better ways to do this!
	switch(opcode)
	{
		case 0x37: // lui
		{
			// We don't need to load it into low bits, then reshift it into high bits... can just read the bits in place!
			// Lower bits are filled with zeros according to standard
			state->x[rd] = inst & 0xfffff000;
			state->pc += 4;
			break;
		}
		case 0x17: // auipc
		{
			// Mirrors the above, but result is imm + offset from pc
			state->x[rd] = state->pc + (inst & 0xfffff000);
			state->pc += 4;
			break;
		}
		case 0x6f: // jal
		{
			// This part seems like it would be much nicer in hardware...
			// The bit order is very strange, [20|10:1|11|19:12]
			// so 31 -> 20 == 11, 30 -> 10 == 20, 20 -> 11 == 9, 19 -> 19 == 0
			// Since right shift doing sign extension is implementation dependent, and
			// this wants sign extension, we do it manually...
			// also, yes, this is correct -- it doesn't set lsb
			uint32_t imm = ((inst & (1 << 31)) >> 11) | ((inst & 0x7fe00000) >> 20) | ((inst & 0x00100000) >> 9) | (inst & 0x000ff000);
			state->x[rd] = state->pc + 4;
			// Two cases: either our machine does sign extension and this is redundant, or it defaults to 0 extension and we need this
			// No machine will default to 1 extension so we're all good
			if(inst & (1 << 31))
			{
				imm |= 0xffe00000;
			}
			state->pc += imm;
			break;
		}
		case 0x67: // jalr
		{
			// This wants us to use a temporary in case the destination register and source register are the same
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t temp = state->pc + 4;
			// Oh yeah we have to sign this one again, but bits are nicer, [11:0], so 31 -> 11 == 20
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			state->pc = (state->x[rs1] + imm) & ~1;
			state->x[rd] = temp;
			break;
		}
		case 0x63: // beq, bne, blt, bge, bltu, bgeu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			// The immediate for jump offset is cursed again, high bits are [12|10:5] and then rd has [4:1|11]
			// 31 -> 12 == 19, 30 -> 10 == 20, 4 -> 4 == 0, 0 -> 11 == -11
			// we have to sign extend again as well
			uint32_t imm = ((inst & (1 << 31)) >> 19) | ((inst & 0x7e000000) >> 20) | (rd & 0x1e) | ((rd & 0x1) << 11);
			if(inst & (1 << 31))
			{
				imm |= 0xffffe000;
			}
			// funct3 (bits 14:12) determines which of the comparisons to do
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // beq
				{
					if(state->x[rs1] == state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x1: // bne
				{
					if(state->x[rs1] != state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x4: // blt (this is signed)
				{
					if((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x5: // bge (this is signed)
				{
					if((int32_t)state->x[rs1] >= (int32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x6: // bltu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				case 0x7: // bgeu (this is unsigned)
				{
					if((uint32_t)state->x[rs1] >= (uint32_t)state->x[rs2]) { state->pc += imm; }
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x03: // lb, lh, lw, lbu, lhu
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			// Same format as jalr
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			uint32_t memOffset = (state->x[rs1] + imm);

			uint32_t funct3 = (inst >> 12) & 0x7;
			uint32_t extra = 0;
			switch(funct3)
			{
				case 0x0: { extra = 0; break; }
				case 0x1: { extra = 1; break; }
				case 0x2: { extra = 3; break; }
				case 0x4: { extra = 0; break; }
				case 0x5: { extra = 1; break; }
			}

			if(memOffset + extra >= memorySize)
			{
				// ERROR
				break;
			}
			uint8_t* basePtr = memory;
			if(memOffset < programSize)
			{
				if(memOffset + extra >= programSize)
				{
					// ERROR, going across regions
					break;
				}
				basePtr = program;
			}
			
			switch(funct3)
			{
				case 0x0: // lb
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 7)) ? loaded | 0xffffff00 : loaded;
					break;
				}
				case 0x1: // lh
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = (loaded & (1 << 15)) ? loaded | 0xffff0000 : loaded;
					break;
				}
				case 0x2: // lw
				{
					state->x[rd] = *(uint32_t*)(basePtr + memOffset);
					break;
				}
				case 0x4: // lbu
				{
					uint8_t loaded = *(uint8_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x000000ff;
					break;
				}
				case 0x5: // lhu
				{
					uint16_t loaded = *(uint16_t*)(basePtr + memOffset);
					state->x[rd] = loaded & 0x0000ffff;
					break;
				}
				// TODO: handle if it isn't one of these? Set trap maybe?
			}
			state->pc += 4;
			break;
		}
		case 0x23: // sb, sh, sw
		{
			// In this one, we reuse rs1 as the memory location (well plus the immediate offset) and we use rs2 as the source
			// This means the immediate is split up again
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			uint32_t imm = ((inst & 0xfe000000) >> 20) | rd;
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // sb
				{
					*(uint8_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				case 0x1: // sh
				{
					*(uint16_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				case 0x2: // sw
				{
					*(uint32_t*)(memory + (state->x[rs1] + imm)) = state->x[rs2];
					break;
				}
				// TODO: handle default?
			}
			state->pc += 4;
			break;
		}
		case 0x13: // addi, slti, sltiu, xori, ori, andi, slli, srli, srai
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t imm = (inst >> 20);
			if(inst & (1 << 31))
			{
				imm |= 0xfffff000;
			}
			// funct3 again
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // addi
				{
					state->x[rd] = state->x[rs1] + imm;
					break;
				}
				case 0x2: // slti
				{
					// I'm pretty sure c standard says true statements always get set to 1 but just to make
					// it clear
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)imm) ? 1 : 0;
					break;
				}
				case 0x3: // sltiu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)imm) ? 1 : 0;
					break;
				}
				case 0x4: // xori
				{
					state->x[rd] = state->x[rs1] ^ imm;
					break;
				}
				case 0x6: // ori
				{
					state->x[rd] = state->x[rs1] | imm;
					break;
				}
				case 0x7: // andi
				{
					state->x[rd] = state->x[rs1] & imm;
					break;
				}
				case 0x1: // slli
				{
					// TODO: these instructions only use the lowest 5 bits of imm, and
					// the standard says the high bits are all 0 (or 1 of them is 1 for srai)
					// I assume it should be illegal operation if that's not the case?
					state->x[rd] = state->x[rs1] << (imm & 0x1f);
					break;
				}
				case 0x5: // srli, srai are differentiated by a 1 in the 30th bit
				{
					uint32_t shamt = imm & 0x1f;
					if(inst & (1 << 30))
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if((state->x[rs1] & (1 << 31)) && shamt)
						{
							// Bit shifts by 32 are undefined by c standard so we actually can't use this which is extremely cringe
							// because it won't work on 0 shift... so we just special case it. 
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						// Don't do sign extension (don't need to do anything special here)
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x33: // add, sub, sll, slt, sltu, xor, srl, sra, or, and
		{
			uint32_t rs1 = (inst >> 15) & 0x1f;
			uint32_t rs2 = (inst >> 20) & 0x1f;
			switch((inst >> 12) & 0x7)
			{
				case 0x0: // add, sub are differentiated again by funct7 (only 1 bit of it tho), inst bit 30
				{
					// Oh and arithmetic overflow is ignored (aka we don't care, and you know what, just use what our implementation does)
					// This isn't 122
					if(inst & (1 << 30)) // add
					{
						state->x[rd] = state->x[rs1] + state->x[rs2];
					}
					else // sub
					{
						state->x[rd] = state->x[rs1] - state->x[rs2];
					}
					break;
				}
				case 0x1: // sll
				{
					// This only cares about the lower 5 bits
					state->x[rd] = state->x[rs1] << (state->x[rs2] & 0x1f);
					break;
				}
				case 0x2: // slt
				{
					state->x[rd] = ((int32_t)state->x[rs1] < (int32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x3: // sltu
				{
					state->x[rd] = ((uint32_t)state->x[rs1] < (uint32_t)state->x[rs2]) ? 1 : 0;
					break;
				}
				case 0x4: // xor
				{
					state->x[rd] = state->x[rs1] ^ state->x[rs2];
					break;
				}
				case 0x5: // srl, sra
				{
					uint32_t shamt = state->x[rs2] & 0x1f;
					if(inst & (1 << 30)) 
					{
						state->x[rd] = (int32_t)(state->x[rs1]) >> shamt;
						if(state->x[rs1] & (1 << 31) && shamt)
						{
							state->x[rd] |= ~0 << (32 - shamt);
						}
					}
					else
					{
						state->x[rd] = (uint32_t)(state->x[rs1]) >> shamt;
					}
					break;
				}
				case 0x6: // or
				{
					state->x[rd] = state->x[rs1] | state->x[rs2];
					break;
				}
				case 0x7: // and
				{
					state->x[rd] = state->x[rs1] & state->x[rs2];
					break;
				}
			}
			state->pc += 4;
			break;
		}
		case 0x0f: // fence, fence.i
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
		case 0x73: // ecall, ebreak, csrrw, csrrs, csrrc, csrrwi, csrrsi, csrrci
		{
			// TODO: do something other than nop?
			state->pc += 4;
			break;
		}
	}

	// We could have written to 0, so just put it back to 0
	if(rd == 0) 
	{
		state->x[rd] = 0;
	}
}

__global__ void kernelExecuteProgram(uint8_t* program, uint8_t* globalMemory, uint32_t memorySize, int32_t argc, uint32_t argv, uint32_t programSize, uint32_t entry)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

	uint8_t* memory = globalMemory + (memorySize * index);

	State state;

	for(int i = 0; i < 32; i++)
	{
		state.x[i] = 0;
	}

	state.pc = entry;

	uint32_t const DONE_ADDRESS_CUDA = 0xfffffff0;
	
	state.x[1] = DONE_ADDRESS_CUDA;
	state.x[2] = argv;

	state.x[10] = argc;
	state.x[11] = argv;

	int count = 0;
	while(count < 1000)
	{
		uint32_t inst = *(uint32_t*)(program + state.pc);
		printf("executing instruction: %08x\n", inst);
	    executeInstruction(&state, inst, memory, program, memorySize, programSize);
		printf("pc = %u\n", state.pc);
		if(state.pc == DONE_ADDRESS_CUDA)
		{
			break;
		}
		count++;
	}
}

// TODO
// STEP 0: Accepts entry point (30 min)
// STEP 1: Cuda program accepts inputs (30 min)
// STEP 2: Cuda program can record (save where we jumped from + where we jumped to) (1 hour)
// STEP 3: Mutation engine (1 hour)

int main(int argc, char** argv)
{	
    if(argc < 3)
	{
        printf("Format: <program file to execute> <entry address as a number in hex> <args to be passed to subject program>");
        return 1;
    }

    uint32_t const MEMORY_SIZE = 4 * 256; // This needs to be 4 byte aligned or bad things happen because cuda memory access rules
	uint32_t const INSTANCE_COUNT = 4;

	// First step: program instructions
	// Reading the program instructions into a buffer
    FILE* programFile = fopen(argv[1], "rb");
    if(!programFile)
	{
        printf("Couldn't open program file \"%s\".\n", argv[1]);
        return 1;
    }
    fseek(programFile, 0, SEEK_END); 
    uint32_t const programSize = ftell(programFile);
    rewind(programFile);
	uint8_t* program = (uint8_t*)malloc(programSize);
    if(!program)
	{
        printf("Failed to allocate enough memory for the instructions for the emulator.\n");
        return 1;
    }
    fread(program, sizeof(uint8_t), programSize, programFile); // We're offset by 4 so we can force 0 addr to be special
	// At this point, host has the program instructions in memory
	uint8_t* deviceProgramImage;
	hipError_t programMallocErrorCode = hipMalloc(&deviceProgramImage, programSize);
	if(programMallocErrorCode != hipSuccess)
	{
		printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(programMallocErrorCode));
		return 1;
	}
	hipMemcpy(deviceProgramImage, program, programSize, hipMemcpyHostToDevice);
	// At this point, device has the program instructions in memory

	// Second step: we need to initialize the state for the processor. This means setting register 0 to all 0s,
	// setting register 1 to the done address (right after last instruction in program), setting register 1 to the top of the stack,
	// setting register 10 to argc, and setting register 11 to argv. To calculate done address and top of stack, we just need to the
	// size of the program and the size of the argument strings, so that means we need to have the input already
	// We also need to set pc, which is constant across instances. All these things we pass when we invoke the kernel
	
	// Third step: input (we're going to base all of our program variability on argv)
	// So we need to produce images of the arguments to send to the device. This is going to reside just above the instance's stack
	// Basically: every instance needs space for initial stuff + some actual stack memory to execute with
	// Nothing is on the stack to start, we pass argc and argv by setting registers 10 and 11
	// So above the stack we have: actual strings, then pointers to them pointed to by argv, then the actual stack
	// So now we allocate the memory images for the program
	uint8_t* memory = (uint8_t*)malloc(MEMORY_SIZE * INSTANCE_COUNT);
	if(!memory)
	{
		printf("Failed to allocate enough memory for the emulator.\n");
		return 1;
	}
	// For now, we're literally just going to pass through arguments from our actual call of this program.
	// So argv[3..] correspond to argv[1..] in the subject program and argv[1] in our program is argv[0] in subject
	int32_t argcSubj = argc - 2;
	uint32_t* argvSubjOffsets = (uint32_t*)malloc(argcSubj * sizeof(uint32_t));
	argvSubjOffsets[0] = strlen(argv[1]) + 1;
	strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[0])), argv[1], argvSubjOffsets[0]);
	for(int32_t i = 1; i < argcSubj; i++)
	{
		// Can't use stpcpy because we need to know size before hand because we are storing "backwards" because we only know
		// Higher address because stack grows down
		uint32_t tempLength = strlen(argv[i + 2]) + 1;
	    argvSubjOffsets[i] = tempLength + argvSubjOffsets[0];
		if(argvSubjOffsets[i] > MEMORY_SIZE)
		{
			printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
			return 1;
		}
		strncpy((char*)(memory + (MEMORY_SIZE - argvSubjOffsets[i])), argv[i + 2], tempLength);
	}
	
	// Still need to copy the pointers to these
	uint32_t argvArrayEnd = argvSubjOffsets[argcSubj - 1];
	argvArrayEnd = argvArrayEnd + ((4 - (argvArrayEnd % 4)) % 4); // Alignment...
	if(argvArrayEnd + (4 * argcSubj) >= MEMORY_SIZE)
	{
		printf("MEMORY_SIZE insufficient to store arg strings for subject program\n");
		return 1;
	}
	for(int32_t i = 0; i < argcSubj; i++)
	{
		// All programs see their memory as offset relative to their own memory chunk so this is ok to copy
		*(uint32_t*)(memory + (MEMORY_SIZE - argvArrayEnd - (4 * (i + 1)))) = MEMORY_SIZE - argvSubjOffsets[argcSubj - i];
	}
	// Now all args are copied to the first instances host memory, so we copy them to all the instances
	uint32_t stackStart = MEMORY_SIZE - (argvArrayEnd + (argcSubj * 4)); // Remember, starting stack pointer value is not usable immediately, dec first, so this ok
	for(uint32_t i = 1; i < INSTANCE_COUNT; i++)
	{
		// Make sure memory size is big enough or problems will happen
		memcpy(memory + ((MEMORY_SIZE * i) + stackStart), memory + stackStart, MEMORY_SIZE - stackStart);
	}
	// Finally can copy all of them to device... a little wasteful, since much of this will be zeroes, but I figure better than many small calls
	// could theoretically seperate these regions of memory but would require complex redirect system on emulator memory system...
	uint8_t* deviceMemoryImage;
    hipError_t mallocMemoryImageError = hipMalloc(&deviceMemoryImage, MEMORY_SIZE * INSTANCE_COUNT);
	if(mallocMemoryImageError != hipSuccess)
	{
		printf("FAILED TO CUDA MALLOC: %s\n", hipGetErrorString(mallocMemoryImageError));
		return 1;
	}
	hipMemcpy(deviceMemoryImage, memory, MEMORY_SIZE * INSTANCE_COUNT, hipMemcpyHostToDevice);
	free(argvSubjOffsets);
	// Should now have both program and memory images on the device

	// TODO fix this later
	dim3 blockDim(INSTANCE_COUNT);
	dim3 gridDim(1);

	uint32_t entryPoint = (uint32_t)strtol(argv[2], NULL, 16);
	
	kernelExecuteProgram<<<gridDim, blockDim>>>(deviceProgramImage, deviceMemoryImage, MEMORY_SIZE, argcSubj, stackStart, programSize, entryPoint);

	hipError_t errorCode = hipPeekAtLastError();
	if(errorCode != hipSuccess)
	{
		printf("FAILED TO LAUNCH KERNEL: %s\n", hipGetErrorString(errorCode));
	}
	hipDeviceSynchronize();

	hipMemcpy(memory, deviceMemoryImage, sizeof(uint8_t) * MEMORY_SIZE * INSTANCE_COUNT, hipMemcpyDeviceToHost);

	uint32_t const BYTES_PER_LINE = 4 * 4;
	for(uint32_t j = 0; j < INSTANCE_COUNT; j++)
	{
		for(uint32_t i = 0; i < MEMORY_SIZE; i += 1)
		{
			if(i % BYTES_PER_LINE == 0)
			{
				printf("\n");
			}
			printf("%02x ", *(uint8_t*)(memory + (j * MEMORY_SIZE) + MEMORY_SIZE - i - 1));
		}
		printf("\n");
	}
	
	hipFree(deviceProgramImage);
	hipFree(deviceMemoryImage);
	
    free(memory);
	free(program);

    return 0;
}
