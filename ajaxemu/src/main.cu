#include "hip/hip_runtime.h"
#include "common.h"
#include "mpi.h"
#include "quad-tree.h"
#include "timing.h"
#include <algorithm>
#include <array>
#include <atomic>
#include <functional>
#include <immintrin.h>
#include <mutex>
#include <queue>
#include <sys/ipc.h>
#include <sys/mman.h>
#include <sys/shm.h>
// #include <thread>
#include <immintrin.h>
#include <unordered_map>
#include <xmmintrin.h>

// stolen from linux, shouldn't be a problem though?
#define unlikely(x) __builtin_expect(!!(x), 0)

#pragma GCC diagnostic ignored "-Wcast-function-type"
#pragma GCC diagnostic ignored "-Wpragmas"
#pragma GCC diagnostic ignored "-Wc++17-extensions"

inline float fastSqrt(const float x) { return _mm_cvtss_f32(_mm_sqrt_ss(_mm_set_ss(x))); }

template<bool careAboutStability>
inline float fastInverseSqrt(float x) {
    // Based on an old Intel post I'd seen a while ago & bookmarked, though the idea is very straightforward
    // http://web.archive.org/web/20140718000055/http://software.intel.com/en-us/articles/interactive-ray-tracing
    // Of course, it's not a great fit for this assignment, so I've changed it considerably to work better for us
    const auto guess = _mm_rsqrt_ss(_mm_set_ss(x));

    if constexpr (!careAboutStability) {
        return _mm_cvtss_f32(guess);
    }

    if (unlikely(x < 10)) {
        const auto muls = _mm_mul_ss(_mm_mul_ss(guess, guess), _mm_set_ss(x));
        const auto half_nr = _mm_mul_ss(_mm_set_ss(0.5f), guess);
        const auto result = _mm_mul_ss(half_nr, _mm_sub_ss(_mm_set_ss(3.0f), muls));
        return _mm_cvtss_f32(result);
    } else {
        return _mm_cvtss_f32(guess);
    }
}

// if this could be properly vectorized, because we can bound the size of near, it'd be possible to do way better
// alas, their (lack of) march settings prevent us from doing that. though I actually haven't checked if
// __attribute__((target("avx2"))) would let us use those intrinsics...
template<bool careAboutStability>
static inline Vec2 computeForceFast(const Particle& target, const Particle& attractor, float cullRadius,
                                    float cullRadius2) {
    auto dir = (attractor.position - target.position);
    const auto dist2 = dir.length2();

    if (dist2 < ((1e-3f) * (1e-3f))) {
        return Vec2(0.0f, 0.0f);
    }
    if (dist2 > cullRadius2) {
        return Vec2(0.0f, 0.0f);
    }

    const float G = 0.01f;
    Vec2 force;
    if (dist2 < (1e-1f * 1e-1f)) {
        // last branch should inform branch here. hopefully gcc doesnt hoist
        dir *= fastInverseSqrt<careAboutStability>(dist2); //(1.0f / sqrt(dist2));
        const auto dist = 1e-1f;                           // gcc will take care of simplifying all of this
        force = dir * attractor.mass * (G / (dist * dist));
        if (dist > cullRadius * 0.75f) {
            float decay = 1.0f - (dist - cullRadius * 0.75f) / (cullRadius * 0.25f);
            force *= decay;
        }
    } else {
        // last branch should inform branch here. hopefully gcc doesnt hoist
        const auto reciprocal = fastInverseSqrt<careAboutStability>(dist2); // (1.0f / sqrt(dist2));
        dir *= reciprocal;
        force = dir * attractor.mass * (G / (dist2));
        if (dist2 > (cullRadius * 0.75f) * (cullRadius * 0.75f)) {
            const auto dist = fastSqrt(dist2);
            float decay = 1.0f - (dist - cullRadius * 0.75f) / (cullRadius * 0.25f);
            force *= decay;
        }
    }
    return force;
}

// Slower version
template<bool usingSharedMemory, bool isKnownDelta>
void simulateStep(QuadTree& quadTree, const Task task, Particle* particles, Particle* newParticles,
                  const StepParameters params) {
    // based on simple-simulator.cpp with edits
    static auto near = std::vector<Particle>();
    float deltaTime;
    if constexpr (isKnownDelta) {
        deltaTime = 0.2f;
    } else {
        deltaTime = params.deltaTime;
    }

    const auto cullRadius = params.cullRadius;

    for (auto i = task.start; i < task.end; i++) {
        const auto& it = particles[i];
        auto force = Vec2(0.0f, 0.0f);
        quadTree.getParticles(near, it.position, params.cullRadius);
        if (!near.empty()) {
            for (const auto& j: near) {
                if ((j.position - it.position).length2() < 0) {
                    __builtin_unreachable();
                }
                force += computeForce(it, j, cullRadius);
            }
        }
        if constexpr (usingSharedMemory) {
            newParticles[i] = updateParticle(it, force, deltaTime);
        } else {
            newParticles[i - task.start] = updateParticle(it, force, deltaTime);
        }
    }
}

// Saves 2 ops
__attribute__((noinline)) static Particle updateParticleFast(const Particle& pi, Vec2 force, float deltaTime) {
    Particle result = pi;
    result.velocity += force * deltaTime;
    result.position += result.velocity * deltaTime;
    return result;
}

__attribute__((target("avx2"))) static inline float hsum(const __m256 reg) {
    float v[8];
    _mm256_storeu_ps(v, reg);
    return ((v[0] + v[1]) + (v[2] + v[3])) + ((v[4] + v[5]) + (v[6] + v[7]));
}

auto near = std::array<Particle, 8192>{}; // sufficient, but larger or smaller doesn't really matter
template<bool usingSharedMemory, bool isKnownDelta, int N, bool careAboutStability>
__attribute__((target("avx2,fma"))) void simulateStep(QuadTree& quadTree, const Task task, Particle* particles,
                                                      Particle* newParticles, const StepParameters params) {
    // based on simple-simulator.cpp with edits
    float deltaTime;
    if constexpr (isKnownDelta) {
        deltaTime = 0.2f;
    } else {
        deltaTime = params.deltaTime;
    }

    constexpr float cullRadius = N * 1.25f;
    constexpr float cullRadius2 = cullRadius * cullRadius;
    static_assert(cullRadius2 == cullRadius * cullRadius);
    constexpr float G = 0.01f;

    for (auto i = task.start; i < task.end; i++) {
        const auto& it = particles[i];
        const auto numParticles = quadTree.getParticles(near.data(), it.position, cullRadius2);

        auto force = Vec2(0.0f, 0.0f);

        // yeah I have 316 homework I don't want to do, how can you tell?
        if (numParticles) {
            const auto& target = it;
            const auto reallyReallyReallyCloseCutoff = _mm256_set1_ps(1e-3f * 1e-3f);
            const auto _cullRadius2 = _mm256_set1_ps(cullRadius2);
            const auto _cullRadius75 = _mm256_set1_ps(cullRadius * 0.75f);
            const auto _cullRadiusInv14 = _mm256_set1_ps(1.0f / (cullRadius * 0.25f));

            const auto targetPosX = _mm256_set1_ps(target.position.x);
            const auto targetPosY = _mm256_set1_ps(target.position.y);

            auto vforceX = _mm256_setzero_ps();
            auto vforceY = _mm256_setzero_ps();

            auto j{0};
            for (; j <= numParticles - 8; j += 8) {
                // for (; j < numParticles - 8; j += 8) {
                // faster for me on my machine but not on ghc :(
                // const auto thisParticlePtr = &near[j];
                //
                // constexpr auto OFFSET_OF_X = offsetof(Particle, position.x);
                // constexpr auto OFFSET_OF_Y = offsetof(Particle, position.y);
                // constexpr auto OFFSET_OF_MASS = offsetof(Particle, mass);
                //
                // const auto vxptr =
                //         _mm256_setr_epi32(OFFSET_OF_X, OFFSET_OF_X + sizeof(Particle),
                //                           OFFSET_OF_X + 2 * sizeof(Particle), OFFSET_OF_X + 3 * sizeof(Particle),
                //                           OFFSET_OF_X + 4 * sizeof(Particle), OFFSET_OF_X + 5 * sizeof(Particle),
                //                           OFFSET_OF_X + 6 * sizeof(Particle), OFFSET_OF_X + 7 * sizeof(Particle));
                //
                // const auto vyptr =
                //         _mm256_setr_epi32(OFFSET_OF_Y, OFFSET_OF_Y + sizeof(Particle),
                //                           OFFSET_OF_Y + 2 * sizeof(Particle), OFFSET_OF_Y + 3 * sizeof(Particle),
                //                           OFFSET_OF_Y + 4 * sizeof(Particle), OFFSET_OF_Y + 5 * sizeof(Particle),
                //                           OFFSET_OF_Y + 6 * sizeof(Particle), OFFSET_OF_Y + 7 * sizeof(Particle));
                //
                // const auto vmassptr =
                //         _mm256_setr_epi32(OFFSET_OF_MASS, OFFSET_OF_MASS + sizeof(Particle),
                //                           OFFSET_OF_MASS + 2 * sizeof(Particle), OFFSET_OF_MASS + 3 *
                //                           sizeof(Particle), OFFSET_OF_MASS + 4 * sizeof(Particle), OFFSET_OF_MASS + 5
                //                           * sizeof(Particle), OFFSET_OF_MASS + 6 * sizeof(Particle), OFFSET_OF_MASS +
                //                           7 * sizeof(Particle));
                //
                // const auto vposX = _mm256_i32gather_ps(reinterpret_cast<const float*>(thisParticlePtr), vxptr, 1);
                //
                // const auto vposY = _mm256_i32gather_ps(reinterpret_cast<const float*>(thisParticlePtr), vyptr, 1);
                //
                // const auto vmass = _mm256_i32gather_ps(reinterpret_cast<const float*>(thisParticlePtr), vmassptr, 1);

                // extremely upsetting that this outperforms the above on ghc
                float xs[8], ys[8], masses[8];
                // #pragma GCC unroll 8
                for (size_t u = 0; u < 8; ++u) {    // 2.1%
                    xs[u] = near[j + u].position.x; // 1.6%
                    ys[u] = near[j + u].position.y; // 0.7%
                    masses[u] = near[j + u].mass;   // 1.5%
                    _mm_prefetch(reinterpret_cast<const char*>(&near[8 + j + u].position.x), _MM_HINT_T0);
                    _mm_prefetch(reinterpret_cast<const char*>(&near[8 + j + u].mass), _MM_HINT_T0);
                }

                const auto vposX = _mm256_loadu_ps(xs);
                const auto vposY = _mm256_loadu_ps(ys);

                auto vdirX = _mm256_sub_ps(vposX, targetPosX);
                auto vdirY = _mm256_sub_ps(vposY, targetPosY);

                const auto vdist2 = _mm256_add_ps(_mm256_mul_ps(vdirX, vdirX), _mm256_mul_ps(vdirY, vdirY));

                // !(dist > cullRadius || dist < 1e-3f)
                const auto mask = _mm256_and_ps(_mm256_cmp_ps(vdist2, reallyReallyReallyCloseCutoff, _CMP_GE_OQ),
                                                _mm256_cmp_ps(vdist2, _cullRadius2, _CMP_LE_OQ));

                if (_mm256_testz_ps(mask, mask)) { // 0.6%
                    continue;
                }

                const auto vinvDist =
                        careAboutStability ? _mm256_div_ps(_mm256_set1_ps(1.0f), _mm256_sqrt_ps(vdist2)) : _mm256_rsqrt_ps(vdist2);
                const auto vdist = _mm256_rcp_ps(vinvDist);
                vdirX = _mm256_mul_ps(vdirX, vinvDist);
                vdirY = _mm256_mul_ps(vdirY, vinvDist);

                // TODO: ideally we reuse vinvDist by transforming it and then squaring (or squaring and transforming)
                const auto dist2norm = _mm256_max_ps(vdist2, _mm256_set1_ps(1e-2f));
                const auto vmass = _mm256_loadu_ps(masses);
                const auto vforce = careAboutStability ? _mm256_div_ps(vmass, dist2norm)
                                                       : _mm256_mul_ps(vmass, _mm256_rcp_ps(dist2norm));
                auto vnewForceX = _mm256_mul_ps(vdirX, vforce);
                auto vnewForceY = _mm256_mul_ps(vdirY, vforce);

                // decay
                const auto vdistnorm = _mm256_max_ps(vdist, _mm256_set1_ps(1e-1f));

                // It'd be nice if we could use this, but the numerical stability that we get from this is such that
                // we're actually notably better than the reference solution. That's a problem because it looks like
                // we're wrong!
                // const auto vdecay = _mm256_fmsub_ps(vdistnorm, _cullRadiusInv14, _mm256_set1_ps(-4.0f));

                // Instead, we use this...
                const auto vdecay = _mm256_sub_ps(_mm256_set1_ps(4.0f), _mm256_mul_ps(vdistnorm, _cullRadiusInv14));
                const auto decayMask = _mm256_cmp_ps(vdistnorm, _cullRadius75, _CMP_GT_OQ);

                // Blends are super fast. The multiplication is the bigger issue here.
                vnewForceX = _mm256_blendv_ps(vnewForceX, _mm256_mul_ps(vnewForceX, vdecay), decayMask);
                vnewForceY = _mm256_blendv_ps(vnewForceY, _mm256_mul_ps(vnewForceY, vdecay), decayMask);

                vforceX = _mm256_add_ps(vforceX, _mm256_and_ps(vnewForceX, mask));
                vforceY = _mm256_add_ps(vforceY, _mm256_and_ps(vnewForceY, mask));
            }

            // plenty fast. dont bother optimizing.
            force.x += hsum(vforceX);
            force.y += hsum(vforceY);

            for (; j < numParticles; j++) {
                const auto& k = near[j];
                const auto& target = it;
                const auto& attractor = k;

                auto dir = (attractor.position - target.position);
                const auto dist2 = dir.length2();

                if (dist2 < ((1e-3f) * (1e-3f))) {
                    continue;
                }
                if (dist2 > cullRadius2) {
                    continue;
                }

                Vec2 newForce;
                if (dist2 < (1e-1f * 1e-1f)) {
                    // last branch should inform branch here. hopefully gcc doesnt hoist
                    dir *= fastInverseSqrt<careAboutStability>(dist2); //(1.0f / sqrt(dist2));
                    const auto dist = 1e-1f;                           // gcc will take care of simplifying all of this
                    newForce = dir * (attractor.mass / (dist * dist));
                    if (dist > cullRadius * 0.75f) {
                        float decay = 1.0f - (dist - cullRadius * 0.75f) / (cullRadius * 0.25f);
                        newForce *= decay;
                    }
                } else {
                    // last branch should inform branch here. hopefully gcc doesnt hoist
                    const auto reciprocal = fastInverseSqrt<careAboutStability>(dist2); // (1.0f / sqrt(dist2));
                    dir *= reciprocal;
                    newForce = dir * (attractor.mass / dist2);
                    if (dist2 > (cullRadius * 0.75f) * (cullRadius * 0.75f)) {
                        const auto dist = fastSqrt(dist2);
                        float decay = 1.0f - (dist - cullRadius * 0.75f) / (cullRadius * 0.25f);
                        newForce *= decay;
                    }
                }

                force += newForce;
            }
        } /*else if constexpr (usingSharedMemory) {
            newParticles[i] = it;
            continue;
        } else {
            newParticles[i - task.start] = it;
            continue;
        }*/
        force *= G;
        if constexpr (usingSharedMemory) {
            newParticles[i] = updateParticleFast(it, force, deltaTime);
        } else {
            newParticles[i - task.start] = updateParticleFast(it, force, deltaTime);
        }
    }
}

// TODO: move to header
using SimulateStepType = void (*)(QuadTree&, const Task, Particle*, Particle*, const StepParameters);
using SpecializationTable = std::unordered_map<int, SimulateStepType>;

SpecializationTable specializedFunctions = {};
SpecializationTable specializedFunctionsDangerous = {};

template<std::size_t N>
struct Initializer {
    static void init(SpecializationTable& safe, SpecializationTable& dangerous) {
        safe[N] = &simulateStep<true, true, N, true>;
        dangerous[N] = &simulateStep<true, true, N, false>;
        Initializer<N - 4>::init(safe, dangerous);
    }
};

template<>
struct Initializer<0> {
    static void init(SpecializationTable& safe, SpecializationTable& dangerous) {
        safe[1] = &simulateStep<true, true, 1, true>;
        dangerous[1] = &simulateStep<true, true, 1, false>;
    }
};

void initializeSpecializedSimulateSteps() {
    Initializer<120>::init(specializedFunctions, specializedFunctionsDangerous);
}

// Textbook bit-interleave. there are like a zillion ways to do this, and since this course has 213 as a prereq, I doubt
// it matters that this isn't original. I'd like to rewrite this but it's not even important to the algorithm and I'd
// like to just have a correct baseline for now. Like, I'm pretty sure 213 even links to Bit Twiddling Hacks?
// Given that pdep/pext exist and we're targeting x86 here (i.e., if the Makefile had march=native, this would almost be
// a builtin) and this is therefore practically a polyfill to overcome an overly restrictive Makefile, I'd hope me copy
// pasting this is fine...
// Anyway, the code is a rip from Knuth's TAOCP
uint64_t interleaveBits(const uint32_t a, const uint32_t b) {
    static const uint64_t masks[] = {0x5555555555555555, 0x3333333333333333, 0x0F0F0F0F0F0F0F0F, 0x00FF00FF00FF00FF,
                                     0x0000FFFF0000FFFF};
    static const uint64_t shifts[] = {1, 2, 4, 8, 16};

    uint64_t result = a | (static_cast<uint64_t>(b) << 32);
    for (int i = 4; i >= 0; --i) {
        result = (result & ~masks[i]) | ((result << shifts[i]) & masks[i]);
    }
    return result;
}

static inline uint32_t toInt(float a, float min, float max) {
    return static_cast<uint32_t>(1000.0f * (a - min) / (max - min));
}

void simulateStepXD(QuadTree& quadTree, const Task task, Particle* particles, Particle* newParticles,
                  const StepParameters params)
{
    // based on simple-simulator.cpp with edits
    static auto near = std::vector<Particle>();
    float deltaTime = params.deltaTime;

    const auto cullRadius = params.cullRadius;

    for (auto i = task.start; i < task.end; i++) {
        const auto& it = particles[i];
        auto force = Vec2(0.0f, 0.0f);
        quadTree.getParticles(near, it.position, params.cullRadius);
        if (!near.empty()) {
            for (const auto& j: near) {
                if ((j.position - it.position).length2() < 0) {
                    __builtin_unreachable();
                }
                force += computeForce(it, j, cullRadius);
            }
        }
		newParticles[i] = updateParticle(it, force, deltaTime);
    }
}

// Ajax solve
void solveAjax(unsigned int const rank, unsigned int const nproc, StartupOptions const& options)
{
	const unsigned int gridEdgeDiv = sqrt(nproc);
	if(gridEdgeDiv * gridEdgeDiv != nproc)
	{
		perror("nproc not square");
		exit(1);
	}
	
	const auto parameters = getBenchmarkStepParams(options.spaceSize);

	std::vector<Particle> particleDump;
	int particleCount;

	//auto particleSortingMap = std::unordered_map<int, int>{};
	
	if(rank == MANAGER_PID)
	{
		loadFromFile(options.inputFile, particleDump);
		particleCount = particleDump.size();

		// for(auto i = 0ul; i < particleDump.size(); i++)
		// {
        //     particleSortingMap[i] = particleDump[i].id;
        // }
	}

	MPI_Bcast(&particleCount, 1, MPI_INT, 0, MPI_COMM_WORLD);
	particleDump.resize(particleCount);
	MPI_Bcast(particleDump.data(), particleCount * sizeof(Particle), MPI_BYTE, 0,
			  MPI_COMM_WORLD);

	// auto particleShmId {0};
    // key_t particleShmKey{1337};

	// Particle* particles {nullptr};

	// // Manager just loads particles to dump and creates shared memory spaces
	// if(rank == MANAGER_PID)
	// {
	// 	loadFromFile(options.inputFile, particleDump);

	// 	const auto SHM_SIZE = particleDump.size() * sizeof(Particle); 

	// 	particleShmId = shmget(particleShmKey, SHM_SIZE, IPC_CREAT | 0666);
	// 	if(particleShmId < 0)
	// 	{
	// 		perror("shmget");
	// 		exit(1);
	// 	}
	// 	particles = static_cast<Particle*>(shmat(particleShmId, nullptr, 0));
    //     if((void*)particles == (void*)-1)
	// 	{
    //         perror("shmat");
    //         exit(1);
    //     }
	// }

	// // Manager tells everyone else where the shared memory is
	// MPI_Bcast(&particleShmId, 1, MPI_INT, MANAGER_PID, MPI_COMM_WORLD);

	// // Manager already did this, now everyone else does -- attaching shared memory
	// if(rank != MANAGER_PID)
	// {
    //     particles = static_cast<Particle*>(shmat(particleShmId, nullptr, 0));
    //     if((void*)particles == (void*)-1)
	// 	{
    //         perror("shmat");
    //         exit(1);
    //     }
	// }

	// Now that everyone has the shared memory, manager just memcpys to it...
	// MPI_Barrier(MPI_COMM_WORLD);
    // if(rank == MANAGER_PID)
	// {
    //     std::memcpy(particles, particleDump.data(), particleDump.size() * sizeof(Particle));
    // }

	//std::cerr << "memcpying particle dump, rank = " << rank << std::endl;

	// ...and then everyone copies it to their particle dump
    // MPI_Barrier(MPI_COMM_WORLD);
    // particleDump.resize(options.numParticles);
    // std::memcpy(particleDump.data(), particles, options.numParticles * sizeof(Particle));

	// We're all set
	MPI_Barrier(MPI_COMM_WORLD);

	Timer totalSimulationTimer;

	const int GRID_UPDATE_REGULARITY = 5;
	
	std::vector<Particle> myAcreParticles;
	std::vector<Particle> myAcreParticlesOut;

	struct AcreBound
	{
		Vec2 min;
		Vec2 max;

		void reset() { min = Vec2(1e30f, 1e30f); max = Vec2(-1e30f, -1e30f); }

		// This is just a quick and easy version, conservative
		static bool interacting(AcreBound const& a, AcreBound const& b, float radius)
			{
				return (a.max.x + radius > b.min.x - radius) && (b.max.x + radius > a.min.x - radius) &&
					(a.max.y + radius > b.min.y - radius) && (b.max.y + radius > a.min.y - radius);
			}
	};

	std::vector<unsigned int> acreCounts(nproc);
	
	AcreBound myAcreBound;
	std::vector<AcreBound> acreBounds(nproc);
	
	std::vector<MPI_Request> sendHandles(nproc);
	std::vector<MPI_Request> recvHandles(nproc);

	unsigned int globalOffset = 0;

	constexpr auto NUMERICAL_INSTABILITY_THRESHOLD = 10; // num iterations before we start caring about instability

	const auto N = parameters.cullRadius / 1.25f;
	const auto specialization = static_cast<int>(std::round(parameters.cullRadius / 1.25f));
    const auto canSpecializeCullRadius = specializedFunctions.find(N) != specializedFunctions.end();
    const auto canSpecializeDeltaTime = parameters.deltaTime == 0.2f;

	//std::cerr << "entering main loop, rank = " << rank << std::endl;
	
	for(unsigned int i = 0; i < options.numIterations; i++)
	{
		MPI_Barrier(MPI_COMM_WORLD);
		if(i % GRID_UPDATE_REGULARITY == 0)
		{

			// Basic idea here is that we create a fixed, square grid that covers all the particles
			// Each worker then is assigned to exactly 1 grid acre (hehe, we're back to acres)
			// This is to simplify communication between threads
			// Each worker then builds a quadtree of the particles inside its acre
		
			// In order to resolve gravitational forces from particles near other acres, we check
			// if any of our particles are nearby other acres and then are ready to ask those other workers
			// for a getParticles at the relevant position

			// As the simulation progresses the worker keeps track of the same set of particles it was
			// assigned, but the acre size can change as the particles move around, thus we will regularly
			// reassign the workers back to acres aligned with the grid

			// NOTE: particleDump is essentially our input buffer and particles is essentially our output buffer
			// for each step. Yes, every worker thread can see every particle, even though for this scheme it's
			// somewhat unnecessary

			// Here we assign workers by id to their acre and then have them build a quadtree of those particles
			// std::vector<size_t> myAcreParticles;
			{
				// Need bounds in order to assign particles to acres...
				// We do a huge amount of redundant work here, especially since
				// we've already had to memcpy this stuff
				Vec2 bmin(1e30f, 1e30f);
				Vec2 bmax(-1e30f, -1e30f);
				for(auto& p : particleDump)
				{
					bmin.x = (bmin.x < p.position.x) ? bmin.x : p.position.x;
					bmin.y = (bmin.y < p.position.y) ? bmin.y : p.position.y;
					bmax.x = (bmax.x > p.position.x) ? bmax.x : p.position.x;
					bmax.y = (bmax.y > p.position.y) ? bmax.y : p.position.y;
				}

				signed char myAcreX = rank % gridEdgeDiv;
				signed char myAcreY = rank / gridEdgeDiv;

				myAcreBound.min = Vec2(((float)myAcreX / (float)gridEdgeDiv) * (bmax.x - bmin.x), ((float)myAcreY / (float)gridEdgeDiv) * (bmax.y - bmin.y));
				myAcreBound.max = Vec2(((float)(myAcreX + 1) / (float)gridEdgeDiv) * (bmax.x - bmin.x), ((float)(myAcreY + 1) / (float)gridEdgeDiv) * (bmax.y - bmin.y));

				myAcreParticles.clear();
				
				for(size_t j = 0; j < particleDump.size(); j++)
				{
					signed char particleAcreX = ((float)gridEdgeDiv * (particleDump[j].position.x - bmin.x)) / ((bmax.x - bmin.x) * 1.0f);
					signed char particleAcreY = ((float)gridEdgeDiv * (particleDump[j].position.y - bmin.y)) / ((bmax.y - bmin.y) * 1.0f);

					if(particleAcreX >= gridEdgeDiv)
					{
						particleAcreX = gridEdgeDiv - 1;
					}
					if(particleAcreY >= gridEdgeDiv)
					{
						particleAcreY = gridEdgeDiv - 1;
					}
					
					if(particleAcreX == myAcreX && particleAcreY == myAcreY)
					{
						// myAcreParticles.push_back(j);
						myAcreParticles.push_back(particleDump[j]);
					}
				}

				myAcreParticlesOut.resize(myAcreParticles.size());

				unsigned int sizeToSend = (unsigned int)(myAcreParticles.size());

				MPI_Allgather(&sizeToSend, sizeof(sizeToSend), MPI_BYTE,
							  acreCounts.data(), sizeof(sizeToSend), MPI_BYTE,
							  MPI_COMM_WORLD);
			}
			//std::cerr << "done setting bounds rank = " << rank << std::endl;
		}
		else
		{
			std::swap(myAcreParticles, myAcreParticlesOut);
		}

		myAcreBound.min = Vec2(1e30f, 1e30f);
		myAcreBound.max = Vec2(-1e30f, -1e30f);
		for(auto& p : myAcreParticles)
		{
			myAcreBound.min.x = myAcreBound.min.x < p.position.x ? myAcreBound.min.x : p.position.x;
			myAcreBound.min.y = myAcreBound.min.y < p.position.y ? myAcreBound.min.y : p.position.y;
			myAcreBound.max.x = myAcreBound.max.x > p.position.x ? myAcreBound.max.x : p.position.x;
			myAcreBound.max.y = myAcreBound.max.y > p.position.y ? myAcreBound.max.y : p.position.y;
		}
		
		// Need to make sure everyone is ready to communicate
		MPI_Barrier(MPI_COMM_WORLD);

		// Everyone calculates their interacting neighbor pairs, then we transfer particles between interacting pairs,
		// then we build everyone's quad trees and they all do their simulation on only their particles
		
		// Everyone needs to know everyone else's acre bounds to see if they interact
		MPI_Allgather(&myAcreBound, sizeof(myAcreBound), MPI_BYTE,
					  acreBounds.data(), sizeof(myAcreBound), MPI_BYTE,
					  MPI_COMM_WORLD);

		MPI_Barrier(MPI_COMM_WORLD);

		std::vector<Particle> localParticles = myAcreParticles;
		
		std::vector<unsigned int> interactors(0);

		unsigned int interactingParticles = 0;

		signed char myAcreX = rank % gridEdgeDiv;
		signed char myAcreY = rank / gridEdgeDiv;

		const int adjust = 2;
		
		// for(int xi = myAcreX - adjust; xi < myAcreX + adjust + 1; xi++)
		// {
		// 	for(int yi = myAcreY - adjust; yi < myAcreY + adjust + 1; yi++)
		// 	{
		// 		if(xi >= 0 && xi < gridEdgeDiv && yi >= 0 && yi < gridEdgeDiv)
		// 		{
		// 			int j = xi + yi * gridEdgeDiv;

		// 			if(j < 0 || j >= (int)nproc || j == rank)
		// 			{

		// 			}
		// 			else if(std::find(interactors.begin(), interactors.end(), j) == interactors.end() && AcreBound::interacting(myAcreBound, acreBounds[j], parameters.cullRadius / 2.0f))
		// 			{
		// 				interactors.emplace_back(j);

		// 				MPI_Isend(myAcreParticles.data(),
		// 						  myAcreParticles.size() * sizeof(Particle),
		// 						  MPI_BYTE,
		// 						  j,
		// 						  0,
		// 						  MPI_COMM_WORLD,
		// 						  &sendHandles[j]);

		// 				interactingParticles += acreCounts[j];
		// 			}
		// 		}
		// 	}
		// }
		
		// for(int k = 0; k < 8; k++)
		// {
		// 	int stride = gridEdgeDiv;
		// 	int j = rank;
		// 	j = rank;
		// 	switch(k)
		// 	{
		// 	case 0: j += -1; break;
		// 	case 1: j += 1; break;
		// 	case 2: j += -stride; break;
		// 	case 3: j += stride; break;
		// 	case 4: j += stride - 1; break;
		// 	case 5: j += stride + 1; break;
		// 	case 6: j += -stride + 1; break;
		// 	case 7: j += -stride - 1; break;
		// 	}
		// 	if(j < 0 || j >= (int)nproc || j == rank)
		// 	{

		// 	}
		// 	else if(std::find(interactors.begin(), interactors.end(), j) == interactors.end() && AcreBound::interacting(myAcreBound, acreBounds[j], parameters.cullRadius / 2.0f))
		// 	{
		// 		interactors.emplace_back(j);

		// 		MPI_Isend(myAcreParticles.data(),
		// 				  myAcreParticles.size() * sizeof(Particle),
		// 				  MPI_BYTE,
		// 				  j,
		// 				  0,
		// 				  MPI_COMM_WORLD,
		// 				  &sendHandles[j]);

		// 		interactingParticles += acreCounts[j];
		// 	}
		// }
		
		for(unsigned int j = 0; j < nproc; j++)
		{
			if(j != rank && AcreBound::interacting(myAcreBound, acreBounds[j], parameters.cullRadius / 2.0f))
			{
				interactors.emplace_back(j);

				MPI_Isend(myAcreParticles.data(),
						  myAcreParticles.size() * sizeof(Particle),
						  MPI_BYTE,
						  j,
						  0,
						  MPI_COMM_WORLD,
						  &sendHandles[j]);

				interactingParticles += acreCounts[j];
			}
		}

		localParticles.resize(myAcreParticles.size() + interactingParticles);

		unsigned int offset = myAcreParticles.size();
		for(unsigned int j = 0; j < interactors.size(); j++)
		{
			auto& actor = interactors[j];
			const auto interactorParticleSize = sizeof(Particle) * acreCounts[actor];
			MPI_Irecv(
				&localParticles[offset],
				interactorParticleSize,
				MPI_BYTE,
				actor,
				0,
				MPI_COMM_WORLD,
				&recvHandles[j]);
			offset += acreCounts[actor];
		}

		MPI_Waitall(interactors.size(), recvHandles.data(), MPI_STATUSES_IGNORE);
		
		// We still need to rebuild our local QuadTree every step, but now each
		// worker only builds the QuadTree containing the particles it has been assigned
		QuadTree tree;
		QuadTree::buildQuadTree(localParticles, tree);

		//std::cerr << "completed building tree, rank = " << rank << std::endl;

		const auto task = Task{0, myAcreParticles.size()};
		//simulateStepXD(tree, task, myAcreParticles.data(), myAcreParticlesOut.data(), parameters);

		const auto& specializationToUse = options.numIterations > NUMERICAL_INSTABILITY_THRESHOLD
			? specializedFunctions[specialization]
			: specializedFunctionsDangerous[specialization];
		specializationToUse(tree, task, myAcreParticles.data(), myAcreParticlesOut.data(), parameters);
	    // else {
		// 	simulateStep<true, false>(tree, task, myAcreParticles.data(), myAcreParticlesOut.data(), parameters);
		// }

		//std::cerr << "completed simulating step, rank = " << rank << std::endl;

		std::vector<int> sizes(nproc), displacements(nproc);
		const auto numParticles = static_cast<std::size_t>(options.numParticles);
		for(auto j = 0ul, offset = 0ul; j < nproc; j++) {
			sizes[j] = acreCounts[j] * sizeof(Particle);
			displacements[j] = static_cast<int>(offset);
			offset += sizes[j];
		}

		if(((i + 1) % GRID_UPDATE_REGULARITY == 0) || (i == options.numIterations - 1))
		{
			if(i == options.numIterations - 1)
			{
				std::sort(myAcreParticlesOut.begin(), myAcreParticlesOut.end(), [](Particle const& a, Particle const& b)
					{ 
						return a.id < b.id; 
					});
			}
			MPI_Barrier(MPI_COMM_WORLD);
			MPI_Allgatherv(myAcreParticlesOut.data(), sizes[rank], MPI_BYTE, particleDump.data(), sizes.data(), displacements.data(),
						   MPI_BYTE, MPI_COMM_WORLD);
		}
		//std::cerr << "completed iter i = " << i << ", rank = " << rank << std::endl;
	}

	printf("All work completed as pid=%d\n", rank);

    MPI_Barrier(MPI_COMM_WORLD);
    printf("Passed barrier as pid=%d\n", rank);
	
    if(rank == MANAGER_PID)
	{		
		std::sort(particleDump.begin(), particleDump.end(), [](Particle const& a, Particle const& b)
			{ 
				return a.id < b.id; 
			});

		printf("total simulation time: %.6fs\n", totalSimulationTimer.elapsed());
		
        std::ofstream f(options.outputFile);
        assert((bool) f && "Cannot open output file");

        f << std::setprecision(9);

		//std::cerr << "printing aprticles to file, part dump size = " << particleDump.size() << std::endl;
		//const auto& p = particleDump[0];
		// std::cerr << p.mass << " " << p.position.x << " " << p.position.y << " " << p.velocity.x << " " << p.velocity.y
		// 		  << std::endl;
        for (auto i = 0ul; i < particleDump.size(); i++)
		{
			//const auto& p = particleDump[particleSortingMap[i]];
			const auto& p = particleDump[i];
            f << p.mass << " " << p.position.x << " " << p.position.y << " " << p.velocity.x << " " << p.velocity.y
              << std::endl;
        }
        assert((bool) f && "Failed to write to output file");

        //shmctl(particleShmId, IPC_RMID, nullptr);
    }
}

template<bool useLoadBalancing>
void solve(const int rank, const int nproc, const StartupOptions& options)
{
    constexpr auto NUMERICAL_INSTABILITY_THRESHOLD = 10; // num iterations before we start caring about instability
    const auto parameters = getBenchmarkStepParams(options.spaceSize);
    std::vector<Particle> particleDump, newParticles;
    auto particleShmId{0};
    auto taskListCounterShmId{0};
    key_t particleShmKey{1337};
    key_t taskListShmKey{7331};
    std::atomic_size_t* taskListIndexPtr{nullptr};
    Particle* particles{nullptr};
    auto particleSortingMap = std::unordered_map<int, int>{};
    const auto N = parameters.cullRadius / 1.25f;
    const auto specialization = static_cast<int>(std::round(parameters.cullRadius / 1.25f));
    const auto canSpecializeCullRadius = specializedFunctions.find(N) != specializedFunctions.end();
    const auto canSpecializeDeltaTime = parameters.deltaTime == 0.2f;

    // Don't bother with load-balanced solutions if it's going to be effectively sequential
    // TODO: can try to do work on pid0
    if (nproc <= 2 && useLoadBalancing) {
        return solve<false>(rank, nproc, options);
    }

    if (rank == MANAGER_PID) {
        loadFromFile(options.inputFile, particleDump); // TODO: check if this gets mapped to huge page, then bench
        auto indexed = std::vector<std::pair<Particle, int>>();

        for (auto i = 0ul; i < particleDump.size(); i++) {
            indexed.emplace_back(particleDump[i], i);
        }

        QuadTree tree;
        QuadTree::buildQuadTree(particleDump, tree);
        const auto swapOrder = (tree.bmax.x - tree.bmin.x) <= (tree.bmax.y - tree.bmin.y);

        // TODO: uncomment
        std::sort(indexed.begin(), indexed.end(), [&tree, swapOrder, N](const auto& l, const auto& r) {
            const auto& a = l.first;
            const auto& b = r.first;

            const auto ax = toInt(N * a.position.x, tree.bmin.x, tree.bmin.x);
            const auto ay = toInt(N * a.position.y, tree.bmin.y, tree.bmin.y);
            const auto bx = toInt(N * b.position.x, tree.bmin.x, tree.bmin.x);
            const auto by = toInt(N * b.position.y, tree.bmin.y, tree.bmin.y);

            const auto az = swapOrder ? interleaveBits(ax, ay) : interleaveBits(ay, ax);
            const auto bz = swapOrder ? interleaveBits(bx, by) : interleaveBits(by, bx);

            return az < bz;
        });

        for (auto i = 0ul; i < particleDump.size(); i++) {
            particleDump[i] = indexed[i].first;
            particleSortingMap[indexed[i].second] = i;
        }

        const auto SHM_SIZE = particleDump.size() * sizeof(Particle); // Size of the shared memory segment

        // TODO: remove
        if constexpr (useLoadBalancing) {
            assert(nproc > 2);
        }
        particleShmId = shmget(particleShmKey, SHM_SIZE, IPC_CREAT | 0666);
        if (particleShmId < 0) {
            perror("shmget");
            exit(1);
        }
        particles = static_cast<Particle*>(shmat(particleShmId, nullptr, 0));
        if ((void*) particles == (void*) -1) {
            perror("shmat");
            exit(1);
        }

        // TODO: change shmsize
        taskListCounterShmId = shmget(taskListShmKey, sizeof(std::atomic_size_t), IPC_CREAT | 0666);
        if (taskListCounterShmId < 0) {
            perror("shmget");
            exit(1);
        }

        taskListIndexPtr = static_cast<std::atomic_size_t*>(shmat(taskListCounterShmId, nullptr, 0));
        if ((void*) taskListIndexPtr == (void*) -1) {
            perror("shmat");
            exit(1);
        }

        *taskListIndexPtr = 0;
    }

    // broadcast the shm ids to everything
    MPI_Bcast(&particleShmId, 1, MPI_INT, MANAGER_PID, MPI_COMM_WORLD);
    MPI_Bcast(&taskListCounterShmId, 1, MPI_INT, MANAGER_PID, MPI_COMM_WORLD);

    if (rank != MANAGER_PID) {
        particles = static_cast<Particle*>(shmat(particleShmId, nullptr, 0));
        if ((void*) particles == (void*) -1) {
            perror("shmat");
            exit(1);
        }

        taskListIndexPtr = static_cast<std::atomic_size_t*>(shmat(taskListCounterShmId, nullptr, 0));
        if ((void*) taskListIndexPtr == (void*) -1) {
            perror("shmat");
            exit(1);
        }
    }

    MPI_Barrier(MPI_COMM_WORLD);

    if (rank == MANAGER_PID) {
        std::memcpy(particles, particleDump.data(), particleDump.size() * sizeof(Particle));
    }

    MPI_Barrier(MPI_COMM_WORLD);
    particleDump.resize(options.numParticles);
    std::memcpy(particleDump.data(), particles, options.numParticles * sizeof(Particle));

    // from the tutorials we're linked in the pdf & told to ref https://hpc-tutorials.llnl.gov/mpi/examples/mpi_heat2D.c
    std::vector<int> sizes(nproc), displacements(nproc);
    if constexpr (!useLoadBalancing) {
        const auto numParticles = static_cast<std::size_t>(options.numParticles);
        for (auto i = 0ul, offset = 0ul, averow = numParticles / nproc, extra = numParticles % nproc;
             i < static_cast<unsigned>(nproc); i++) {
            sizes[i] = (averow + (i < extra)) * sizeof(Particle);
            displacements[i] = static_cast<int>(offset);
            offset += sizes[i];
        }
        newParticles.resize(sizes[rank] / sizeof(Particle));
    }

    Timer totalSimulationTimer;

    // TODO: Ideally, replace this with a lockless queue and don't have to worry about a manager. That's part of where
    // we end up losing so much speed.
    //
    for (int i = 0; i < options.numIterations; i++) {
        if constexpr (useLoadBalancing) {
            constexpr auto SCALING_FACTOR = 4;

            const auto numParticles = static_cast<std::size_t>(options.numParticles);
            const auto chunkSize = std::min(numParticles, numParticles / (SCALING_FACTOR * nproc));
            const auto totalChunks = numParticles / chunkSize;

            while (true) {
                QuadTree tree;
                QuadTree::buildQuadTree(particleDump, tree);

                const auto taskIdx = std::atomic_fetch_add(taskListIndexPtr, 1);

                if (taskIdx > totalChunks) {
                    break;
                }

                const auto task = Task{taskIdx * chunkSize, std::min(numParticles, (taskIdx + 1) * chunkSize)};

                if (canSpecializeDeltaTime && canSpecializeCullRadius) {
                    const auto& specializationToUse = options.numIterations > NUMERICAL_INSTABILITY_THRESHOLD
                                                              ? specializedFunctions[specialization]
                                                              : specializedFunctionsDangerous[specialization];
                    specializationToUse(tree, task, /* in */ particleDump.data(),
                                        /* out */ particles, parameters);
                } else {
                    simulateStep<true, false>(tree, task, /* in */ particleDump.data(), /* out */ particles,
                                              parameters);
                }
            }

            // Post-iteration, set particleDump = particles
            MPI_Barrier(MPI_COMM_WORLD);
            std::atomic_store(taskListIndexPtr, 0);
            std::memcpy(particleDump.data(), particles, particleDump.size() * sizeof(Particle));
            MPI_Barrier(MPI_COMM_WORLD);
            // end if constexpr
        } else {
            MPI_Barrier(MPI_COMM_WORLD);
            QuadTree tree;
            QuadTree::buildQuadTree(particleDump, tree);
            const auto task = Task{static_cast<std::size_t>(displacements[rank] / sizeof(Particle)),
                                   (displacements[rank] + sizes[rank]) / sizeof(Particle)};
            if (canSpecializeCullRadius && canSpecializeDeltaTime) {
                const auto& specializationToUse = options.numIterations > NUMERICAL_INSTABILITY_THRESHOLD
                                                          ? specializedFunctions[specialization]
                                                          : specializedFunctionsDangerous[specialization];
                specializationToUse(tree, task, /* in */ particleDump.data(), /* out */ particles, parameters);
            } else {
                simulateStep<true, false>(tree, task, /* in */ particleDump.data(), /* out */ particles, parameters);
            }
            MPI_Barrier(MPI_COMM_WORLD);
            // We also have a version that uses allgatherv on our repository if you would want to see that.
            std::memcpy(particleDump.data(), particles, particleDump.size() * sizeof(Particle));
        }
    }

    printf("All work completed as pid=%d\n", rank);

    MPI_Barrier(MPI_COMM_WORLD);
    printf("Passed barrier as pid=%d\n", rank);

    if (rank == MANAGER_PID) {
        printf("total simulation time: %.6fs\n", totalSimulationTimer.elapsed());

        std::ofstream f(options.outputFile);
        assert((bool) f && "Cannot open output file");

        f << std::setprecision(9);
        for (auto i = 0ul; i < particleDump.size(); i++) {
            const auto& p = particleDump[particleSortingMap[i]];
            f << p.mass << " " << p.position.x << " " << p.position.y << " " << p.velocity.x << " " << p.velocity.y
              << std::endl;
        }
        assert((bool) f && "Failed to write to output file");

        shmctl(particleShmId, IPC_RMID, nullptr);
        shmctl(taskListCounterShmId, IPC_RMID, nullptr);
    }
}

int main(int argc, char* argv[])
{
    int pid, nproc;
    initializeMPI(argc, argv, pid, nproc);
    initializeSpecializedSimulateSteps();
	
    const auto options = parseOptions(argc, argv);

	if(options.numParticles <= 100000)
	{
		if(options.loadBalance)
		{
			solve<true>(pid, nproc, options);
		}
		else
		{
			solve<false>(pid, nproc, options);
		}
	}
	else
	{
		solveAjax(pid, nproc, options);
	}
	
    // if(options.loadBalance)
	// {
    //     solve<true>(pid, nproc, options);
    // }
	// else
	// {
    //     solve<false>(pid, nproc, options);
    // }

	finalizeMPI();
	
    return 0;
}
